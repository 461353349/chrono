#include "hip/hip_runtime.h"
/*
 * printToFile.cu
 *
 *  Created on: Mar 2, 2015
 *      Author: Arman Pazouki
 */
#include <string.h>
#include <stdio.h>
#include <sstream>
#include <fstream>
#include <thrust/reduce.h>
#include "chrono_fsi/printToFile.cuh"
#include "chrono_fsi/custom_cutil_math.h"
#include "chrono_fsi/SPHCudaUtils.h"
using namespace std;

//*******************************************************************************************************************************
void PrintCartesianData_MidLine(const thrust::host_vector<Real4>& rho_Pres_CartH,
                                const thrust::host_vector<Real4>& vel_VelMag_CartH,
                                const int3& cartesianGridDims,
                                const SimParams& paramsH) {
  int3 gridCenter = mI3(cartesianGridDims.x / 2, cartesianGridDims.y / 2, cartesianGridDims.z / 2);
  stringstream midLineProfile;
  for (int k = 0; k < cartesianGridDims.z; k++) {
    // Assuming flow in x Direction, walls on Z direction, periodic on y direction
    int index = (cartesianGridDims.x * cartesianGridDims.y) * k + cartesianGridDims.x * gridCenter.y + gridCenter.x;
    Real3 v = mR3(vel_VelMag_CartH[index]);
    Real3 rp = mR3(rho_Pres_CartH[index]);
    //		midLineProfile << v.x << ", " << v.y << ", " << v.z << ", " << length(v) << ", " << rp.x << ", " << rp.y
    //<<
    // endl;
    midLineProfile << v.x << ", ";
  }
  midLineProfile << endl;
  static int count = 0;
  ofstream midLineData;
  if (count == 0) {
    midLineData.open("MidLineData.txt");
  } else {
    midLineData.open("MidLineData.txt", ios::app);
  }
  count++;
  midLineData << midLineProfile.str();
  midLineData.close();
}

//*******************************************************************************************************************************
void PrintToFile_SPH(const thrust::device_vector<Real3>& posRadD,
                     const thrust::device_vector<Real3>& velMasD,
                     const thrust::device_vector<Real4>& rhoPresMuD,
                     const thrust::host_vector<int4>& referenceArray,

                     const SimParams paramsH,
                     const Real realTime,
                     int tStep,
                     int stepSave,
                     const std::string& out_dir) {
  thrust::host_vector<Real3> posRadH = posRadD;
  thrust::host_vector<Real3> velMasH = velMasD;
  thrust::host_vector<Real4> rhoPresMuH = rhoPresMuD;

  int tStepsPovFiles = stepSave;  // 25;//1000;//2000;
  if (tStep % tStepsPovFiles == 0) {
    //#ifdef _WIN32
    //			system("mkdir povFiles");
    //#else
    //			system("mkdir -p povFiles");
    //#endif
    if (tStep / tStepsPovFiles == 0) {
      const string rmCmd = string("rm ") + out_dir + string("/*.csv");
      system(rmCmd.c_str());
    }
    char fileCounter[5];
    int dumNumChar = sprintf(fileCounter, "%d", int(tStep / tStepsPovFiles));

    //*****************************************************
    const string nameFluid = out_dir + string("/fluid") + string(fileCounter) + string(".csv");

    ofstream fileNameFluidParticles;
    fileNameFluidParticles.open(nameFluid);
    stringstream ssFluidParticles;
    ssFluidParticles << "x, y, z, vx, vy, vz, rho, p, mu, type\n";
    for (int i = referenceArray[0].x; i < referenceArray[0].y; i++) {
      Real3 pos = posRadH[i];
      Real3 vel = velMasH[i];
      Real4 rP = rhoPresMuH[i];
      Real velMag = length(vel);
      ssFluidParticles << pos.x << ", " << pos.y << ", " << pos.z << ", " << vel.x << ", " << vel.y << ", " << vel.z
                       << ", " << rP.x << ", " << rP.y << ", " << rP.z << ", " << rP.w << ", " << endl;
    }
    fileNameFluidParticles << ssFluidParticles.str();
    fileNameFluidParticles.close();
    //*****************************************************
    const string nameBoundary = out_dir + string("/boundary") + string(fileCounter) + string(".csv");

    //    ofstream fileNameBoundaries;
    //    fileNameBoundaries.open(nameBoundary);
    //    stringstream ssBoundary;
    //    for (int i = referenceArray[1].x; i < referenceArray[1].y; i++) {
    //      Real3 pos = posRadH[i];
    //      Real3 vel = velMasH[i];
    //      Real4 rP = rhoPresMuH[i];
    //      Real velMag = length(vel);
    //      ssBoundary << pos.x << ", " << pos.y << ", " << pos.z << ", " << vel.x << ", " << vel.y << ", " << vel.z <<
    //      ", "
    //                 << velMag << ",
    //                              "<< rP.x<<",
    //          "<< rP.y<<", "<< rP.w<<", "<<endl;
    //    }
    //    fileNameBoundaries << ssBoundary.str();
    //    fileNameBoundaries.close();
    //*****************************************************
    const string nameFluidBoundaries = out_dir + string("/fluid_boundary") + string(fileCounter) + string(".csv");

    ofstream fileNameFluidBoundaries;
    fileNameFluidBoundaries.open(nameFluidBoundaries);
    stringstream ssFluidBoundaryParticles;
    //		ssFluidBoundaryParticles.precision(20);
    ssFluidBoundaryParticles << "x, y, z, vx, vy, vz, rho, p, mu, type\n";

    for (int i = referenceArray[0].x; i < referenceArray[1].y; i++) {
      Real3 pos = posRadH[i];
      Real3 vel = velMasH[i];
      Real4 rP = rhoPresMuH[i];
      Real velMag = length(vel);
      // if (pos.y > .0002 && pos.y < .0008)
      ssFluidBoundaryParticles << pos.x << ", " << pos.y << ", " << pos.z << ", " << vel.x << ", " << vel.y << ", "
                               << vel.z << ", " << rP.x << ", " << rP.y << ", " << rP.z << ", " << rP.w << endl;
    }
    fileNameFluidBoundaries << ssFluidBoundaryParticles.str();
    fileNameFluidBoundaries.close();
    //*****************************************************
    const string nameBCE = out_dir + string("/BCE") + string(fileCounter) + string(".csv");

    ofstream fileNameBCE;
    fileNameBCE.open(nameBCE);
    stringstream ssBCE;
    //		ssFluidBoundaryParticles.precision(20);
    ssBCE << "x, y, z, vx, vy, vz, rho, p, mu, type\n";

    int refSize = referenceArray.size();
    if (refSize > 2) {
      for (int i = referenceArray[2].x; i < referenceArray[refSize - 1].y; i++) {
        Real3 pos = posRadH[i];
        Real3 vel = velMasH[i];
        Real4 rP = rhoPresMuH[i];
        Real velMag = length(vel);
        // if (pos.y > .0002 && pos.y < .0008)
        ssBCE << pos.x << ", " << pos.y << ", " << pos.z << ", " << vel.x << ", " << vel.y << ", " << vel.z << ", "
              << velMag << ", " << rP.x << ", " << rP.y << ", " << rP.z << ", " << rP.w << endl;
      }
    }
    fileNameBCE << ssBCE.str();
    fileNameBCE.close();
    //*****************************************************
  }
  posRadH.clear();
  velMasH.clear();
  rhoPresMuH.clear();
}

//*******************************************************************************************************************************

void PrintToFile(const thrust::device_vector<Real3>& posRadD,
                 const thrust::device_vector<Real3>& velMasD,
                 const thrust::device_vector<Real4>& rhoPresMuD,
                 const thrust::host_vector<int4>& referenceArray,
                 const SimParams paramsH,
                 Real realTime,
                 int tStep,
                 int stepSave,
                 const string& out_dir) {
  // print fluid stuff
  PrintToFile_SPH(posRadD, velMasD, rhoPresMuD, referenceArray, paramsH, realTime, tStep, stepSave, out_dir);
}
//*******************************************************************************************************************************
// to be implemented
void PrintToFileCartesian() {
  //  // ######## the commented sections need to be fixed. you need cartesian data by calling SphSystemGpu.MapSPH_ToGrid
  //  ////////-+++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++//comcom
  //  ofstream fileNameCartesianTotal;
  //  thrust::host_vector<Real4> rho_Pres_CartH(1);
  //  thrust::host_vector<Real4> vel_VelMag_CartH(1);
  //  Real resolution = 2 * paramsH.HSML;
  //  int3 cartesianGridDims;
  //  int tStepCartesianTotal = 1000000;
  //  int tStepCartesianSlice = 100000;
  //  int tStepPoiseuilleProf = 1000;  // tStepCartesianSlice;
  //
  //  int stepCalcCartesian = min(tStepCartesianTotal, tStepCartesianSlice);
  //  stepCalcCartesian = min(stepCalcCartesian, tStepPoiseuilleProf);
  //
  //  if (tStep % stepCalcCartesian == 0) {
  //    MapSPH_ToGrid(resolution, cartesianGridDims, rho_Pres_CartH, vel_VelMag_CartH, posRadD, velMasD, rhoPresMuD,
  //                  referenceArray[referenceArray.size() - 1].y, paramsH);
  //  }
  //  if (tStep % tStepCartesianTotal == 0) {
  //    if (tStep / tStepCartesianTotal == 0) {
  //      fileNameCartesianTotal.open("dataCartesianTotal.txt");
  //      fileNameCartesianTotal << "variables = \"x\", \"y\", \"z\", \"Vx\", \"Vy\", \"Vz\", \"Velocity
  //          Magnitude\", \"Rho\", \"Pressure\"\n";
  //    } else {
  //      fileNameCartesianTotal.open("dataCartesianTotal.txt", ios::app);
  //    }
  //    fileNameCartesianTotal << "zone I = " << cartesianGridDims.x << ", J = " << cartesianGridDims.y
  //                           << ", K =
  //                              "<<cartesianGridDims.z<<endl;
  //        stringstream ssCartesianTotal;
  //    for (int k = 0; k < cartesianGridDims.z; k++) {
  //      for (int j = 0; j < cartesianGridDims.y; j++) {
  //        for (int i = 0; i < cartesianGridDims.x; i++) {
  //          int index = i + j * cartesianGridDims.x + k * cartesianGridDims.x * cartesianGridDims.y;
  //          Real3 gridNodeLoc = resolution * mR3(i, j, k) + paramsH.worldOrigin;
  //          ssCartesianTotal << gridNodeLoc.x << ", " << gridNodeLoc.y << ", " << gridNodeLoc.z
  //                           << ",
  //                              "<<
  //                              vel_VelMag_CartH[index]
  //                                  .x
  //                           << ", " << vel_VelMag_CartH[index].y
  //                           << ",
  //                              "<<
  //                              vel_VelMag_CartH[index]
  //                                  .z
  //                           << ", " << vel_VelMag_CartH[index].w << ", " << rho_Pres_CartH[index].x << ", "
  //                           << rho_Pres_CartH[index].y << endl;
  //        }
  //      }
  //    }
  //    fileNameCartesianTotal << ssCartesianTotal.str();
  //    fileNameCartesianTotal.close();
  //  }
  //  ////////-+++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++ //comcom
  //  ofstream fileNameCartesianMidplane;
  //  if (tStep % tStepCartesianSlice == 0) {
  //    if (tStep / tStepCartesianSlice == 0) {
  //      fileNameCartesianMidplane.open("dataCartesianMidplane.txt");
  //      fileNameCartesianMidplane << "variables = \"x\", \"z\", \"Vx\", \"Vy\", \"Vz\", \"Velocity
  //          Magnitude\",
  //	\"Rho\", \"Pressure\"\n";
  //    } else {
  //      fileNameCartesianMidplane.open("dataCartesianMidplane.txt", ios::app);
  //    }
  //    fileNameCartesianMidplane << "zone I = " << cartesianGridDims.x << ", J = " << cartesianGridDims.z << "\n";
  //    int j = cartesianGridDims.y / 2;
  //    stringstream ssCartesianMidplane;
  //    for (int k = 0; k < cartesianGridDims.z; k++) {
  //      for (int i = 0; i < cartesianGridDims.x; i++) {
  //        int index = i + j * cartesianGridDims.x + k * cartesianGridDims.x * cartesianGridDims.y;
  //        Real3 gridNodeLoc = resolution * mR3(i, j, k) + paramsH.worldOrigin;
  //        ssCartesianMidplane << gridNodeLoc.x << ", " << gridNodeLoc.z << ", " << vel_VelMag_CartH[index].x
  //                            << ",
  //                               "<<
  //                               vel_VelMag_CartH[index]
  //                                   .y
  //                            << ", " << vel_VelMag_CartH[index].z << ", " << vel_VelMag_CartH[index].w << ", "
  //                            << rho_Pres_CartH[index].x << ", " << rho_Pres_CartH[index].y << endl;
  //      }
  //    }
  //    fileNameCartesianMidplane << ssCartesianMidplane.str();
  //    fileNameCartesianMidplane.close();
  //  }
  //  rho_Pres_CartH.clear();
  //  ////////-+++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++comcom
  //  ofstream fileVelocityProfPoiseuille;
  //  if (tStep % tStepPoiseuilleProf == 0) {
  //    if (tStep / tStepPoiseuilleProf == 0) {
  //      fileVelocityProfPoiseuille.open("dataVelProfile.txt");
  //      fileVelocityProfPoiseuille << "variables = \"Z(m)\", \"Vx(m/s)\"\n";
  //
  //    } else {
  //      fileVelocityProfPoiseuille.open("dataVelProfile.txt", ios::app);
  //    }
  //    fileVelocityProfPoiseuille << "zone T=\"t = " << realTime << "\"" endl;
  //    stringstream ssVelocityProfPoiseuille;
  //    int j = cartesianGridDims.y / 2;
  //    int i = cartesianGridDims.x / 2;
  //    for (int k = 0; k < cartesianGridDims.z; k++) {
  //      int index = i + j * cartesianGridDims.x + k * cartesianGridDims.x * cartesianGridDims.y;
  //      Real3 gridNodeLoc = resolution * mR3(i, j, k) + paramsH.worldOrigin;
  //      if (gridNodeLoc.z > 1 * paramsH.sizeScale && gridNodeLoc.z < 2 * paramsH.sizeScale) {
  //        ssVelocityProfPoiseuille << gridNodeLoc.z << ", " << vel_VelMag_CartH[index].x << endl;
  //      }
  //    }
  //    fileVelocityProfPoiseuille << ssVelocityProfPoiseuille.str();
  //    fileVelocityProfPoiseuille.close();
  //  }
  //  vel_VelMag_CartH.clear();
  //  //////////-+++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++comcom
}
