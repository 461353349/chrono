#include "hip/hip_runtime.h"
/*
 * SDKCollisionSystem.cu
 *
 *  Created on: Mar 2, 2013
 *      Author: Arman Pazouki
 */
#include <stdexcept>
#include <thrust/sort.h>
#include "chrono_fsi/SDKCollisionSystem.cuh"


//#include "extraOptionalFunctions.cuh"
//#include "SDKCollisionSystemAdditional.cuh"

/**
 * @brief calcGridHash
 * @details  See SDKCollisionSystem.cuh
 */
__device__ int3 calcGridPos(Real3 p) {
	int3 gridPos;
	gridPos.x = floor((p.x - paramsD.worldOrigin.x) / paramsD.cellSize.x);
	gridPos.y = floor((p.y - paramsD.worldOrigin.y) / paramsD.cellSize.y);
	gridPos.z = floor((p.z - paramsD.worldOrigin.z) / paramsD.cellSize.z);
	return gridPos;
}

/**
 * @brief calcGridHash
 * @details  See SDKCollisionSystem.cuh
 */
__device__ uint calcGridHash(int3 gridPos) {
	gridPos.x -= ((gridPos.x >= paramsD.gridSize.x) ? paramsD.gridSize.x : 0);
	gridPos.y -= ((gridPos.y >= paramsD.gridSize.y) ? paramsD.gridSize.y : 0);
	gridPos.z -= ((gridPos.z >= paramsD.gridSize.z) ? paramsD.gridSize.z : 0);

	gridPos.x += ((gridPos.x < 0) ? paramsD.gridSize.x : 0);
	gridPos.y += ((gridPos.y < 0) ? paramsD.gridSize.y : 0);
	gridPos.z += ((gridPos.z < 0) ? paramsD.gridSize.z : 0);

	return __umul24(__umul24(gridPos.z, paramsD.gridSize.y), paramsD.gridSize.x)
			+ __umul24(gridPos.y, paramsD.gridSize.x) + gridPos.x;
}

/**
 * @brief calcGridHash
 * @details  See SDKCollisionSystem.cuh
 */
__device__ inline Real4 DifVelocityRho(Real3& dist3, Real& d, Real3 posRadA, Real3 posRadB,
		Real3& velMasA, Real3& vel_XSPH_A, Real3& velMasB,
		Real3& vel_XSPH_B, Real4& rhoPresMuA,
		Real4& rhoPresMuB, Real multViscosity) {
	Real3 gradW = GradW(dist3);

	// Real vAB_Dot_rAB = dot(velMasA - velMasB, dist3);

	//	//*** Artificial viscosity type 1.1
	//	Real alpha = .001;
	//	Real c_ab = 10 * paramsD.v_Max; //Ma = .1;//sqrt(7.0f * 10000 / ((rhoPresMuA.x + rhoPresMuB.x) / 2.0f));
	//	//Real h = paramsD.HSML;
	//	Real rho = .5f * (rhoPresMuA.x + rhoPresMuB.x);
	//	Real nu = alpha * paramsD.HSML * c_ab / rho;

	//	//*** Artificial viscosity type 1.2
	//	Real nu = 22.8f * paramsD.mu0 / 2.0f / (rhoPresMuA.x * rhoPresMuB.x);
	//	Real3 derivV = -paramsD.markerMass * (
	//		rhoPresMuA.y / (rhoPresMuA.x * rhoPresMuA.x) + rhoPresMuB.y / (rhoPresMuB.x * rhoPresMuB.x)
	//		- nu * vAB_Dot_rAB / ( d * d + paramsD.epsMinMarkersDis * paramsD.HSML * paramsD.HSML )
	//		) * gradW;
	//	return mR4(derivV,
	//		rhoPresMuA.x * paramsD.markerMass / rhoPresMuB.x * dot(vel_XSPH_A - vel_XSPH_B, gradW));

	//*** Artificial viscosity type 2
	Real rAB_Dot_GradW = dot(dist3, gradW);
	Real rAB_Dot_GradW_OverDist = rAB_Dot_GradW
			/ (d * d + paramsD.epsMinMarkersDis * paramsD.HSML * paramsD.HSML);
	Real3 derivV = -paramsD.markerMass
			* (rhoPresMuA.y / (rhoPresMuA.x * rhoPresMuA.x)
					+ rhoPresMuB.y / (rhoPresMuB.x * rhoPresMuB.x)) * gradW
			+ paramsD.markerMass * (8.0f * multViscosity) * paramsD.mu0
					* pow(rhoPresMuA.x + rhoPresMuB.x, Real(-2))
					* rAB_Dot_GradW_OverDist * (velMasA - velMasB);
	Real derivRho = rhoPresMuA.x * paramsD.markerMass / rhoPresMuB.x
			* dot(vel_XSPH_A - vel_XSPH_B, gradW);
	//	Real zeta = 0;//.05;//.1;
	//	Real derivRho = rhoPresMuA.x * paramsD.markerMass * invrhoPresMuBx * (dot(vel_XSPH_A - vel_XSPH_B, gradW)
	//			+ zeta * paramsD.HSML * (10 * paramsD.v_Max) * 2 * (rhoPresMuB.x / rhoPresMuA.x - 1) *
	// rAB_Dot_GradW_OverDist
	//			);

	//--------------------------------
	// Ferrari Modification
	derivRho = paramsD.markerMass * dot(vel_XSPH_A - vel_XSPH_B, gradW);
	Real cA = FerrariCi(rhoPresMuA.x);
	Real cB = FerrariCi(rhoPresMuB.x);
	derivRho -= rAB_Dot_GradW / (d + paramsD.epsMinMarkersDis * paramsD.HSML) * max(cA, cB) / rhoPresMuB.x * (rhoPresMuB.x - rhoPresMuA.x);

	//--------------------------------
	return mR4(derivV, derivRho);

	//	//*** Artificial viscosity type 1.3
	//	Real rAB_Dot_GradW = dot(dist3, gradW);
	//	Real3 derivV = -paramsD.markerMass * (rhoPresMuA.y / (rhoPresMuA.x * rhoPresMuA.x) + rhoPresMuB.y / (rhoPresMuB.x *
	// rhoPresMuB.x)) * gradW
	//		+ paramsD.markerMass / (rhoPresMuA.x * rhoPresMuB.x) * 2.0f * paramsD.mu0 * rAB_Dot_GradW / ( d * d +
	// paramsD.epsMinMarkersDis * paramsD.HSML * paramsD.HSML ) * (velMasA - velMasB);
	//	return mR4(derivV,
	//		rhoPresMuA.x * paramsD.markerMass / rhoPresMuB.x * dot(vel_XSPH_A - vel_XSPH_B, gradW));
}
//--------------------------------------------------------------------------------------------------------------------------------
__device__ inline Real3 DifVelocity_SSI_DEM(const Real3& dist3, const Real& d,
		const Real3& velMasA, const Real3& velMasB) {
	// printf("** DifVelocity_SSI_DEM\n");
	Real l = paramsD.MULT_INITSPACE * paramsD.HSML - d;  // penetration distance
	if (l < 0) {
		return mR3(0);
	}
	Real kS = .00006; // 6;//3; //50; //1000.0; //392400.0;	//spring. 50 worked almost fine. I am using 30 to be
					  // sure!
	Real kD = 40; // 20;//40.0;//20.0; //420.0;				//damping coef. // 40 is good don't change it.
	Real3 n = dist3 / d;  // unit vector B to A
	Real m_eff = 0.5 * paramsD.markerMass; //(mA * mB) / (mA + mB);
	Real3 force = (/*pow(paramsD.sizeScale, Real(3)) * */kS * l
			- kD * m_eff * dot(velMasA - velMasB, n)) * n; // relative velocity at contact is simply assumed as the relative vel of the centers. If you are
														   // updating the rotation, this should be modified.
	return force / paramsD.markerMass;  // return dV/dT same as SPH
}
//--------------------------------------------------------------------------------------------------------------------------------
__device__ inline Real3 DifVelocity_SSI_Lubrication(const Real3& dist3,
		const Real& d, const Real3& velMasA, const Real3& velMasB) {
	// printf("** DifVelocity_SSI_Lubrication\n");
	Real Delta_c = paramsD.HSML;
	Real s = d - paramsD.MULT_INITSPACE * paramsD.HSML;
	if (s > Delta_c)
		return mR3(0);

	Real Delta_i = .1 * Delta_c;
	Real mult = 0;
	if (s > Delta_i) {
		mult = 1 / s - 1 / Delta_c;
	} else {
		mult = 1 / Delta_i - 1 / Delta_c;
	}
	Real3 n = dist3 / d;  // unit vector B to A
	Real3 force = -(mult * 1.5 * PI * paramsD.mu0 * paramsD.HSML * paramsD.HSML)
			* dot(velMasA - velMasB, n) * n;
	return force / paramsD.markerMass;
}
//--------------------------------------------------------------------------------------------------------------------------------
// collide a particle against all other particles in a given cell
__device__ Real3 deltaVShare(int3 gridPos, uint index, Real3 posRadA,
		Real3 velMasA, Real4 rhoPresMuA, Real3* sortedPosRad,
		Real3* sortedVelMas, Real4* sortedRhoPreMu, uint* cellStart,
		uint* cellEnd) {
	uint gridHash = calcGridHash(gridPos);
	// get start of bucket for this cell
	Real3 deltaV = mR3(0.0f);

	uint startIndex = FETCH(cellStart, gridHash);
	if (startIndex != 0xffffffff) {  // cell is not empty
		// iterate over particles in this cell
		uint endIndex = FETCH(cellEnd, gridHash);

		for (uint j = startIndex; j < endIndex; j++) {
			if (j != index) {  // check not colliding with self
				Real3 posRadB = FETCH(sortedPosRad, j);
				Real3 dist3 = Distance(posRadA, posRadB);
				Real d = length(dist3);
				if (d > RESOLUTION_LENGTH_MULT * paramsD.HSML)
					continue;
				Real4 rhoPresMuB = FETCH(sortedRhoPreMu, j);
				if (rhoPresMuB.w > -.1)
					continue; //# B must be fluid (A was checked originally and it is fluid at this point), accoring to
				// colagrossi (2003), the other phase (i.e. rigid) should not be considered)
				Real multRho = 2.0f / (rhoPresMuA.x + rhoPresMuB.x);
				Real3 velMasB = FETCH(sortedVelMas, j);
				deltaV += paramsD.markerMass * (velMasB - velMasA) * W3(d)
						* multRho;
			}
		}
	}
	return deltaV;
}
//--------------------------------------------------------------------------------------------------------------------------------
// collide a particle against all other particles in a given cell
// Arman : revisit equation 10 of tech report, is it only on fluid or it is on all markers
__device__ void BCE_modification_Share(
		Real3& sumVW,
		Real& sumWAll,
		Real3& sumRhoRW,
		Real& sumPW,
		Real& sumWFluid,
		int& isAffectedV, int& isAffectedP, int3 gridPos,
		Real3 posRadA, Real3* sortedPosRad, Real3* sortedVelMas,
		Real4* sortedRhoPreMu, uint* cellStart, uint* cellEnd) {
	uint gridHash = calcGridHash(gridPos);
	// get start of bucket for this cell
	uint startIndex = FETCH(cellStart, gridHash);
	if (startIndex != 0xffffffff) {  // cell is not empty
		// iterate over particles in this cell
		uint endIndex = FETCH(cellEnd, gridHash);

		for (uint j = startIndex; j < endIndex; j++) {
			Real3 posRadB = FETCH(sortedPosRad, j);
			Real3 dist3 = Distance(posRadA, posRadB);
			Real d = length(dist3);
			Real4 rhoPresMuB = FETCH(sortedRhoPreMu, j);
			if (d > RESOLUTION_LENGTH_MULT * paramsD.HSML || rhoPresMuB.w > -.1)
				continue;

			Real Wd = W3(d);
			Real WdOvRho = Wd / rhoPresMuB.x;
			isAffectedV = 1;
			Real3 velMasB = FETCH(sortedVelMas, j);
			sumVW += velMasB * WdOvRho;
			sumWAll += WdOvRho;

			isAffectedP = 1;
			sumRhoRW += rhoPresMuB.x * dist3 * WdOvRho;
			sumPW += rhoPresMuB.y * WdOvRho;
			sumWFluid += WdOvRho;
		}
	}
}
//--------------------------------------------------------------------------------------------------------------------------------
// modify pressure for body force
__device__ __inline__ void modifyPressure(Real4& rhoPresMuB,
		const Real3& dist3Alpha) {
	// body force in x direction
	rhoPresMuB.y =
			(dist3Alpha.x > 0.5 * paramsD.boxDims.x) ?
					(rhoPresMuB.y - paramsD.deltaPress.x) : rhoPresMuB.y;
	rhoPresMuB.y =
			(dist3Alpha.x < -0.5 * paramsD.boxDims.x) ?
					(rhoPresMuB.y + paramsD.deltaPress.x) : rhoPresMuB.y;
	// body force in x direction
	rhoPresMuB.y =
			(dist3Alpha.y > 0.5 * paramsD.boxDims.y) ?
					(rhoPresMuB.y - paramsD.deltaPress.y) : rhoPresMuB.y;
	rhoPresMuB.y =
			(dist3Alpha.y < -0.5 * paramsD.boxDims.y) ?
					(rhoPresMuB.y + paramsD.deltaPress.y) : rhoPresMuB.y;
	// body force in x direction
	rhoPresMuB.y =
			(dist3Alpha.z > 0.5 * paramsD.boxDims.z) ?
					(rhoPresMuB.y - paramsD.deltaPress.z) : rhoPresMuB.y;
	rhoPresMuB.y =
			(dist3Alpha.z < -0.5 * paramsD.boxDims.z) ?
					(rhoPresMuB.y + paramsD.deltaPress.z) : rhoPresMuB.y;
}
//--------------------------------------------------------------------------------------------------------------------------------
// collide a particle against all other particles in a given cell
__device__ Real4 collideCell(int3 gridPos, uint index, Real3 posRadA,
		Real3 velMasA, Real3 vel_XSPH_A, Real4 rhoPresMuA, Real3* sortedPosRad,
		Real3* sortedVelMas, Real3* vel_XSPH_Sorted_D, Real4* sortedRhoPreMu,
		Real3* velMas_ModifiedBCE, Real4* rhoPreMu_ModifiedBCE, uint* gridMarkerIndex,
		uint* cellStart, uint* cellEnd) {

	uint gridHash = calcGridHash(gridPos);
	// get start of bucket for this cell
	Real4 derivVelRho = mR4(0);

	uint startIndex = FETCH(cellStart, gridHash);
	if (startIndex == 0xffffffff) { // cell is not empty
		return derivVelRho;
	}
	// iterate over particles in this cell
	uint endIndex = FETCH(cellEnd, gridHash);

	for (uint j = startIndex; j < endIndex; j++) {
		if (j != index) {  // check not colliding with self
			Real3 posRadB = FETCH(sortedPosRad, j);
			Real3 dist3Alpha = posRadA - posRadB;
//			Real3 dist3 = Distance(posRadA, posRadB);
			Real3 dist3 = Modify_Local_PosB(posRadB, posRadA);
			Real d = length(dist3);
			if (d > RESOLUTION_LENGTH_MULT * paramsD.HSML)
				continue;

			Real4 rhoPresMuB = FETCH(sortedRhoPreMu, j);
//			// old version. When rigid-rigid contact used to be handled within fluid
//			if ((fabs(rhoPresMuB.w - rhoPresMuA.w) < .1)
//					&& rhoPresMuA.w > -.1) {
//				continue;
//			}
			if (rhoPresMuA.w > -.1 && rhoPresMuB.w > -.1) { // no rigid-rigid force
				continue;
			}

			modifyPressure(rhoPresMuB, dist3Alpha);
			Real3 velMasB = FETCH(sortedVelMas, j);
			if (rhoPresMuB.w > -.1) {
				int bceIndexB = gridMarkerIndex[j] - (numObjectsD.numFluidMarkers);
				if (!(bceIndexB >= 0 && bceIndexB < numObjectsD.numBoundaryMarkers + numObjectsD.numRigid_SphMarkers)) {
					printf("Error! bceIndex out of bound, collideD !\n");
				}
				rhoPresMuB = rhoPreMu_ModifiedBCE[bceIndexB];
				velMasB = velMas_ModifiedBCE[bceIndexB];
			}
			Real multViscosit = 1;
			Real4 derivVelRhoAB = mR4(0.0f);
			Real3 vel_XSPH_B = FETCH(vel_XSPH_Sorted_D, j);
			derivVelRhoAB = DifVelocityRho(dist3, d, posRadA, posRadB, velMasA, vel_XSPH_A,
					velMasB, vel_XSPH_B, rhoPresMuA, rhoPresMuB,
					multViscosit);
			derivVelRho += derivVelRhoAB;
		}
	}

	// ff1
	//	if (rhoPresMuA.w > 0) printf("force value %f %f %f\n", 1e20*derivV.x, 1e20*derivV.y, 1e20*derivV.z);
	return derivVelRho;
} //--------------------------------------------------------------------------------------------------------------------------------
// collide a particle against all other particles in a given cell
__device__ __inline__ void stressCell(Real3& devS3, Real3& volS3, int3 gridPos,
		uint index, Real3 posRadA, Real3 velMasA, Real4 rhoPresMuA,
		Real3* sortedPosRad, Real3* sortedVelMas, Real4* sortedRhoPreMu,
		uint* cellStart, uint* cellEnd) {

	uint gridHash = calcGridHash(gridPos);
	// get start of bucket for this cell
	Real3 derivV = mR3(0.0f);

	uint startIndex = FETCH(cellStart, gridHash);
	if (startIndex != 0xffffffff) {  // cell is not empty
		// iterate over particles in this cell
		uint endIndex = FETCH(cellEnd, gridHash);

		for (uint j = startIndex; j < endIndex; j++) {
			if (j != index) {  // check not colliding with self
				Real3 posRadB = FETCH(sortedPosRad, j);
				Real3 dist3Alpha = posRadA - posRadB;
				Real3 dist3 = Distance(posRadA, posRadB);
				Real d = length(dist3);
				if (d > RESOLUTION_LENGTH_MULT * paramsD.HSML)
					continue;

				Real3 velMasB = FETCH(sortedVelMas, j);
				Real4 rhoPresMuB = FETCH(sortedRhoPreMu, j);

				Real3 vr = velMasB - velMasA;
				Real3 gradW = GradW(dist3);

				// Randles and Libersky, 1996
				devS3 += -paramsD.mu0 * paramsD.markerMass / rhoPresMuB.x
						*
						mR3(vr.x * gradW.y + vr.y * gradW.x,
								vr.x * gradW.z + vr.z * gradW.x,
								vr.y * gradW.z + vr.z * gradW.y);
				volS3 += -paramsD.mu0 * paramsD.markerMass / rhoPresMuB.x * 4.0
						/ 3.0
						* mR3(vr.x * gradW.x, vr.y * gradW.y, vr.z * gradW.z);
			}
		}
	}
}
//--------------------------------------------------------------------------------------------------------------------------------
// collide a particle against all other particles in a given cell
__device__ void collideCellDensityReInit(Real& densityShare, Real& denominator,
		int3 gridPos, uint index, Real3 posRadA, Real3* sortedPosRad,
		Real3* sortedVelMas, Real4* sortedRhoPreMu, uint* cellStart,
		uint* cellEnd) {

	//?c2 printf("grid pos %d %d %d \n", gridPos.x, gridPos.y, gridPos.z);
	uint gridHash = calcGridHash(gridPos);
	// get start of bucket for this cell
	Real densityShare2 = 0.0f;
	Real denominator2 = 0.0f;

	uint startIndex = FETCH(cellStart, gridHash);
	if (startIndex != 0xffffffff) {  // cell is not empty
		// iterate over particles in this cell
		uint endIndex = FETCH(cellEnd, gridHash);

		for (uint j = startIndex; j < endIndex; j++) {
			if (j != index) {  // check not colliding with self
				Real3 posRadB = FETCH(sortedPosRad, j);
				Real4 rhoPreMuB = FETCH(sortedRhoPreMu, j);
				Real3 dist3 = Distance(posRadA, posRadB);
				Real d = length(dist3);
				if (d > RESOLUTION_LENGTH_MULT * paramsD.HSML)
					continue;
				Real partialDensity = paramsD.markerMass * W3(d); // optimize it ?$
				densityShare2 += partialDensity;
				denominator2 += partialDensity / rhoPreMuB.x;
				// if (fabs(W3(d)) < .00000001) {printf("good evening, distance %f %f %f\n", dist3.x, dist3.y, dist3.z);
				// printf("posRadA %f %f %f, posRadB, %f %f %f\n", posRadA.x, posRadA.y, posRadA.z, posRadB.x, posRadB.y,
				// posRadB.z);
				//}
			}
		}
	}
	densityShare += densityShare2;
	denominator += denominator2;
}
//--------------------------------------------------------------------------------------------------------------------------------
// collide a particle against all other particles in a given cell
__device__ void projectTheClosestFluidMarker(Real3& distRhoPress, int3 gridPos,
		uint index, Real3 posRadA, Real3* sortedPosRad, Real4* sortedRhoPreMu,
		uint* cellStart, uint* cellEnd) {

	//?c2 printf("grid pos %d %d %d \n", gridPos.x, gridPos.y, gridPos.z);
	uint gridHash = calcGridHash(gridPos);
	// get start of bucket for this cell
	uint startIndex = FETCH(cellStart, gridHash);
	if (startIndex != 0xffffffff) {  // cell is not empty
		// iterate over particles in this cell
		uint endIndex = FETCH(cellEnd, gridHash);

		for (uint j = startIndex; j < endIndex; j++) {
			if (j == index)
				continue;
			Real4 rhoPreMuB = FETCH(sortedRhoPreMu, j);
			if (rhoPreMuB.w > -.1)
				continue;  // we don't care about the closest non-fluid marker
			Real3 posRadB = FETCH(sortedPosRad, j);
			Real3 dist3 = Distance(posRadA, posRadB);
			Real d = length(dist3);
			if (distRhoPress.x > d) {
				distRhoPress = mR3(d, rhoPreMuB.x, rhoPreMuB.y);
			}
		}
	}
}
//--------------------------------------------------------------------------------------------------------------------------------
// collide a particle against all other particles in a given cell
__device__ void calcOnCartesianShare(Real3& v_share, Real4& rp_share,
		int3 gridPos, Real3 gridNodePos3, Real3* sortedPosRad,
		Real3* sortedVelMas, Real4* sortedRhoPreMu, uint* cellStart,
		uint* cellEnd) {

	//?c2 printf("grid pos %d %d %d \n", gridPos.x, gridPos.y, gridPos.z);
	uint gridHash = calcGridHash(gridPos);
	// get start of bucket for this cell
	uint startIndex = FETCH(cellStart, gridHash);
	if (startIndex != 0xffffffff) {  // cell is not empty
		// iterate over particles in this cell
		uint endIndex = FETCH(cellEnd, gridHash);

		for (uint j = startIndex; j < endIndex; j++) {
			Real3 posRadB = FETCH(sortedPosRad, j);
			Real3 velMasB = FETCH(sortedVelMas, j);
			Real4 rhoPreMuB = FETCH(sortedRhoPreMu, j);
			Real3 dist3 = Distance(gridNodePos3, posRadB);
			Real d = length(dist3);
			Real mult = paramsD.markerMass / rhoPreMuB.x * W3(d);
			v_share += mult * velMasB;  // optimize it ?$
			rp_share += mult * mR4(rhoPreMuB.x, rhoPreMuB.y, 0, 0);
		}
	}
}

/**
 * @brief calcHashD
 * @details
 * 		 1. Get particle index. Determine by the block and thread we are in.
 * 		 2. From x,y,z position determine which bin it is in.
 * 		 3. Calculate hash from bin index.
 * 		 4. Store hash and particle index associated with it.
 *
 * @param gridMarkerHash
 * @param gridMarkerIndex
 * @param posRad
 * @param numAllMarkers
 */
__global__ void calcHashD(uint* gridMarkerHash,   // output
		uint* gridMarkerIndex,  // output
		Real3* posRad,          // input: positions
		uint numAllMarkers, volatile bool *isErrorD) {

	/* Calculate the index of where the particle is stored in posRad. */
	uint index = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;
	if (index >= numAllMarkers)
		return;

	Real3 p = posRad[index];

	if (!(isfinite(p.x) && isfinite(p.y) && isfinite(p.z))) {
		printf("Error! particle position is NAN: thrown from SDKCollisionSystem.cu, calcHashD !\n");
		*isErrorD = true;
		return;
	}

	/* Check particle is inside the domain. */
	Real3 boxCorner = paramsD.worldOrigin;
	if (p.x < boxCorner.x || p.y < boxCorner.y || p.z < boxCorner.z) {
		printf("Out of Min Boundary, point %f %f %f, boundary min: %f %f %f. Thrown from SDKCollisionSystem.cu, calcHashD !\n",
				p.x, p.y, p.z,boxCorner.x, boxCorner.y, boxCorner.z);
		*isErrorD = true;
		return;
	}
	boxCorner = paramsD.worldOrigin + paramsD.boxDims;
	if (p.x > boxCorner.x || p.y > boxCorner.y || p.z > boxCorner.z) {
		printf(
				"Out of max Boundary, point %f %f %f, boundary max: %f %f %f. Thrown from SDKCollisionSystem.cu, calcHashD !\n",
				p.x, p.y, p.z, boxCorner.x, boxCorner.y, boxCorner.z);
		*isErrorD = true;
		return;
	}

	/* Get x,y,z bin index in grid */
	int3 gridPos = calcGridPos(p);
	/* Calculate a hash from the bin index */
	uint hash = calcGridHash(gridPos);

	/* Store grid hash */
	gridMarkerHash[index] = hash;
	/* Store particle index associated to the hash we stored in gridMarkerHash */
	gridMarkerIndex[index] = index;
}

/**
 * @brief reorderDataAndFindCellStartD
 * @details See SDKCollisionSystem.cuh for more info
 */
__global__ void reorderDataAndFindCellStartD(uint* cellStart, // output: cell start index
		uint* cellEnd,        // output: cell end index
		Real3* sortedPosRad,  // output: sorted positions
		Real3* sortedVelMas,  // output: sorted velocities
		Real4* sortedRhoPreMu, uint* gridMarkerHash, // input: sorted grid hashes
		uint* gridMarkerIndex,      // input: sorted particle indices
		uint* mapOriginalToSorted, // mapOriginalToSorted[originalIndex] = originalIndex
		Real3* oldPosRad,           // input: sorted position array
		Real3* oldVelMas,           // input: sorted velocity array
		Real4* oldRhoPreMu, uint numAllMarkers) {
	extern __shared__ uint sharedHash[];  // blockSize + 1 elements
	/* Get the particle index the current thread is supposed to be looking at. */
	uint index = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;
	uint hash;
	/* handle case when no. of particles not multiple of block size */
	if (index < numAllMarkers) {
		hash = gridMarkerHash[index];
		/* Load hash data into shared memory so that we can look at neighboring particle's hash
		 * value without loading two hash values per thread
		 */
		sharedHash[threadIdx.x + 1] = hash;

		if (index > 0 && threadIdx.x == 0) {
			/* first thread in block must load neighbor particle hash */
			sharedHash[0] = gridMarkerHash[index - 1];
		}
	}

	__syncthreads();

	if (index < numAllMarkers) {
		/* If this particle has a different cell index to the previous particle then it must be
		 * the first particle in the cell, so store the index of this particle in the cell. As it
		 * isn't the first particle, it must also be the cell end of the previous particle's cell
		 */
		if (index == 0 || hash != sharedHash[threadIdx.x]) {
			cellStart[hash] = index;
			if (index > 0)
				cellEnd[sharedHash[threadIdx.x]] = index;
		}

		if (index == numAllMarkers - 1) {
			cellEnd[hash] = index + 1;
		}

		/* Now use the sorted index to reorder the pos and vel data */
		uint originalIndex = gridMarkerIndex[index];  // map sorted to original
		mapOriginalToSorted[index] = index;	// will be sorted outside. Alternatively, you could have mapOriginalToSorted[originalIndex] = index; without need to sort. But that is not thread safe
		Real3 posRad = FETCH(oldPosRad, originalIndex); // macro does either global read or texture fetch
		Real3 velMas = FETCH(oldVelMas, originalIndex); // see particles_kernel.cuh
		Real4 rhoPreMu = FETCH(oldRhoPreMu, originalIndex);

		if (!(isfinite(posRad.x) && isfinite(posRad.y)
				&& isfinite(posRad.z))) {
			printf("Error! particle position is NAN: thrown from SDKCollisionSystem.cu, reorderDataAndFindCellStartD !\n");
		}
		if (!(isfinite(velMas.x) && isfinite(velMas.y)
				&& isfinite(velMas.z))) {
			printf("Error! particle velocity is NAN: thrown from SDKCollisionSystem.cu, reorderDataAndFindCellStartD !\n");
		}
		if (!(isfinite(rhoPreMu.x) && isfinite(rhoPreMu.y)
				&& isfinite(rhoPreMu.z) && isfinite(rhoPreMu.w))) {
			printf("Error! particle rhoPreMu is NAN: thrown from SDKCollisionSystem.cu, reorderDataAndFindCellStartD !\n");
		}
		sortedPosRad[index] = posRad;
		sortedVelMas[index] = velMas;
		sortedRhoPreMu[index] = rhoPreMu;
	}
}
//--------------------------------------------------------------------------------------------------------------------------------
__global__ void newVel_XSPH_D(Real3* vel_XSPH_Sorted_D,  // output: new velocity
		Real3* sortedPosRad,       // input: sorted positions
		Real3* sortedVelMas,       // input: sorted velocities
		Real4* sortedRhoPreMu, uint* gridMarkerIndex, // input: sorted particle indices
		uint* cellStart, uint* cellEnd, uint numAllMarkers, volatile bool *isErrorD) {
	uint index = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;
	if (index >= numAllMarkers)
		return;

	// read particle data from sorted arrays

	Real4 rhoPreMuA = FETCH(sortedRhoPreMu, index);
	Real3 velMasA = FETCH(sortedVelMas, index);
	if (rhoPreMuA.w > -0.1) { // v_XSPH is calculated only for fluid markers. Keep unchanged if not fluid.
		vel_XSPH_Sorted_D[index] = velMasA;
		return;
	}

	Real3 posRadA = FETCH(sortedPosRad, index);
	Real3 deltaV = mR3(0);

	// get address in grid
	int3 gridPos = calcGridPos(posRadA);

	/// if (gridPos.x == paramsD.gridSize.x-1) printf("****aha %d %d\n", gridPos.x, paramsD.gridSize.x);

	// examine neighbouring cells
	for (int z = -1; z <= 1; z++) {
		for (int y = -1; y <= 1; y++) {
			for (int x = -1; x <= 1; x++) {
				int3 neighbourPos = gridPos + mI3(x, y, z);
				deltaV += deltaVShare(neighbourPos, index, posRadA, velMasA,
						rhoPreMuA, sortedPosRad, sortedVelMas, sortedRhoPreMu,
						cellStart, cellEnd);
			}
		}
	}
	//   // write new velocity back to original unsorted location
	// sortedVel_XSPH[index] = velMasA + paramsD.EPS_XSPH * deltaV;

	// write new velocity back to original unsorted location
	// uint originalIndex = gridMarkerIndex[index];
	Real3 vXSPH = velMasA + paramsD.EPS_XSPH * deltaV;
	if (!(isfinite(vXSPH.x) && isfinite(vXSPH.y)
			&& isfinite(vXSPH.z))) {
		printf("Error! particle vXSPH is NAN: thrown from SDKCollisionSystem.cu, newVel_XSPH_D !\n");
		*isErrorD = true;
	}
	vel_XSPH_Sorted_D[index] = vXSPH;
}
//--------------------------------------------------------------------------------------------------------------------------------
__global__ void new_BCE_VelocityPressure(
		Real3* velMas_ModifiedBCE,    		// input: sorted velocities
		Real4* rhoPreMu_ModifiedBCE,  		// input: sorted velocities
		Real3* sortedPosRad,                // input: sorted positions
		Real3* sortedVelMas,                // input: sorted velocities
		Real4* sortedRhoPreMu,
		uint* cellStart,
		uint* cellEnd,
		uint* mapOriginalToSorted,
		Real3* bceAcc,
		int2 updatePortion,
		volatile bool *isErrorD) {
	uint bceIndex = blockIdx.x * blockDim.x + threadIdx.x;
	uint sphIndex = bceIndex + updatePortion.x; // updatePortion = [start, end] index of the update portion
	if (sphIndex >= updatePortion.y) {
		return;
	}
	uint idA = mapOriginalToSorted[sphIndex];
	Real4 rhoPreMuA = FETCH(sortedRhoPreMu, idA);
	Real3 posRadA = FETCH(sortedPosRad, idA);
	Real3 velMasA = FETCH(sortedVelMas, idA);
	int isAffectedV = 0;
	int isAffectedP = 0;


	Real3 sumVW = mR3(0);
	Real sumWAll = 0;
	Real3 sumRhoRW = mR3(0);
	Real sumPW = 0;
	Real sumWFluid = 0;

	// get address in grid
	int3 gridPos = calcGridPos(posRadA);

	/// if (gridPos.x == paramsD.gridSize.x-1) printf("****aha %d %d\n", gridPos.x, paramsD.gridSize.x);

	// examine neighbouring cells
	for (int z = -1; z <= 1; z++) {
		for (int y = -1; y <= 1; y++) {
			for (int x = -1; x <= 1; x++) {
				int3 neighbourPos = gridPos + mI3(x, y, z);
				BCE_modification_Share(sumVW, sumWAll, sumRhoRW, sumPW, sumWFluid, isAffectedV, isAffectedP,
						neighbourPos, posRadA, sortedPosRad,
						sortedVelMas, sortedRhoPreMu, cellStart, cellEnd);
			}
		}
	}

	if (isAffectedV) {
		Real3 modifiedBCE_v = 2 * velMasA - sumVW / sumWAll;
		velMas_ModifiedBCE[bceIndex] = modifiedBCE_v;
	}
	if (isAffectedP) {
		// pressure
		Real3 a3 = mR3(0);
		if (fabs(rhoPreMuA.w) > 0) {  // rigid BCE
			int rigidBceIndex = sphIndex - numObjectsD.startRigidMarkers;
			if (rigidBceIndex < 0 || rigidBceIndex >= numObjectsD.numRigid_SphMarkers) {
				printf("Error! marker index out of bound: thrown from SDKCollisionSystem.cu, new_BCE_VelocityPressure !\n");
				*isErrorD = true;
				return;
			}
			a3 = bceAcc[rigidBceIndex];
		}
		Real pressure = (sumPW + dot(paramsD.gravity - a3, sumRhoRW))
				/ sumWFluid;  //(in fact:  (paramsD.gravity -
		// aW), but aW for moving rigids
		// is hard to calc. Assume aW is
		// zero for now
		Real density = InvEos(pressure);
		rhoPreMu_ModifiedBCE[bceIndex] = mR4(density, pressure, rhoPreMuA.z,
				rhoPreMuA.w);
	}
}
//--------------------------------------------------------------------------------------------------------------------------------
__global__ void collideD(Real4* sortedDerivVelRho_fsi_D,  // output: new velocity
		Real3* sortedPosRad,  // input: sorted positions
		Real3* sortedVelMas,  // input: sorted velocities
		Real3* vel_XSPH_Sorted_D, Real4* sortedRhoPreMu,
		Real3* velMas_ModifiedBCE, Real4* rhoPreMu_ModifiedBCE, uint* gridMarkerIndex,
		uint* cellStart, uint* cellEnd, uint numAllMarkers, volatile bool *isErrorD) {

	uint index = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;
	if (index >= numAllMarkers)
		return;

	// read particle data from sorted arrays
	Real3 posRadA = FETCH(sortedPosRad, index);
	Real3 velMasA = FETCH(sortedVelMas, index);
	Real4 rhoPreMuA = FETCH(sortedRhoPreMu, index);


	// *** comment these couple of lines since we don't want the force on the rigid (or boundary) be influenced by ADAMi
	// *** method since it would cause large forces. ADAMI method is used only to calculate forces on the fluid markers (A)
	// *** near the boundary or rigid (B).
//	if (rhoPreMuA.w > -.1) {
//		int bceIndex = gridMarkerIndex[index] - (numObjectsD.numFluidMarkers);
//		if (!(bceIndex >= 0 && bceIndex < numObjectsD.numBoundaryMarkers + numObjectsD.numRigid_SphMarkers)) {
//			printf("Error! bceIndex out of bound, collideD !\n");
//			*isErrorD = true;
//		}
//		rhoPreMuA = rhoPreMu_ModifiedBCE[bceIndex];
//		velMasA = velMas_ModifiedBCE[bceIndex];
//	}

//	uint originalIndex = gridMarkerIndex[index];
	Real3 vel_XSPH_A = vel_XSPH_Sorted_D[index];
	Real4 derivVelRho = sortedDerivVelRho_fsi_D[index];

	// get address in grid
	int3 gridPos = calcGridPos(posRadA);

	// examine neighbouring cells
	for (int x = -1; x <= 1; x++) {
		for (int y = -1; y <= 1; y++) {
			for (int z = -1; z <= 1; z++) {
				derivVelRho += collideCell(gridPos + mI3(x, y, z), index,
						posRadA, velMasA, vel_XSPH_A, rhoPreMuA, sortedPosRad,
						sortedVelMas, vel_XSPH_Sorted_D, sortedRhoPreMu,
						velMas_ModifiedBCE, rhoPreMu_ModifiedBCE, gridMarkerIndex,
						cellStart, cellEnd);
			}
		}
	}

	// write new velocity back to original unsorted location
	// *** let's tweak a little bit :)
	if (!(isfinite(derivVelRho.x) && isfinite(derivVelRho.y)
			&& isfinite(derivVelRho.z) )) {
		printf("Error! particle derivVel is NAN: thrown from SDKCollisionSystem.cu, collideD !\n");
		*isErrorD = true;
	}
	if (!(isfinite(derivVelRho.w))) {
		printf("Error! particle derivRho is NAN: thrown from SDKCollisionSystem.cu, collideD !\n");
		*isErrorD = true;
	}
	sortedDerivVelRho_fsi_D[index] = derivVelRho;
}
//--------------------------------------------------------------------------------------------------------------------------------
// calculate particles stresses
__global__ void CalcBCE_Stresses_kernel(Real3* devStressD, Real3* volStressD,
		Real3* sortedPosRad, Real3* sortedVelMas, Real4* sortedRhoPreMu,
		uint* mapOriginalToSorted, uint* cellStart, uint* cellEnd, int numBCE) {
	uint index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= numBCE) {
		return;
	}
	// Arman take care of this
	uint BCE_Index = index
			+ min(numObjectsD.startRigidMarkers, numObjectsD.startRigidMarkers); // updatePortion = [start, end] index of the update portion
	uint originalIndex = mapOriginalToSorted[BCE_Index]; // index in the sorted array

	// read particle data from sorted arrays
	Real3 posRadA = FETCH(sortedPosRad, originalIndex);
	Real3 velMasA = FETCH(sortedVelMas, originalIndex);
	Real4 rhoPreMuA = FETCH(sortedRhoPreMu, originalIndex);

	// get address in grid
	int3 gridPos = calcGridPos(posRadA);

	Real3 devS3 = mR3(0);
	Real3 volS3 = mR3(0);

	// examine neighbouring cells
	for (int x = -1; x <= 1; x++) {
		for (int y = -1; y <= 1; y++) {
			for (int z = -1; z <= 1; z++) {
				stressCell(devS3, volS3, gridPos + mI3(x, y, z), originalIndex,
						posRadA, velMasA, rhoPreMuA, sortedPosRad, sortedVelMas,
						sortedRhoPreMu, cellStart, cellEnd);
			}
		}
	}

	devStressD[index] = devS3;
	volStressD[index] = volS3;
}
//--------------------------------------------------------------------------------------------------------------------------------
// calculate particles stresses
__global__ void CalcBCE_MainStresses_kernel(Real4* mainStressD,
		Real3* devStressD, Real3* volStressD, int numBCE) {
	uint index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= numBCE) {
		return;
	}

	Real3 devS3 = devStressD[index];
	Real3 volS3 = volStressD[index];
	Real4 mainS3 = mR4(0);
	mainS3.w = sqrt(
			.5
					* (pow(volS3.x - volS3.y, Real(2))
							+ pow(volS3.x - volS3.z, Real(2))
							+ pow(volS3.y - volS3.z, Real(2))
							+ 6
									* (devS3.x * devS3.x + devS3.y * devS3.y
											+ devS3.z * devS3.z)));
	mainStressD[index] = mainS3;
}
//--------------------------------------------------------------------------------------------------------------------------------
// without normalization
__global__ void ReCalcDensityD_F1(Real4* dummySortedRhoPreMu, Real3* sortedPosRad, Real3* sortedVelMas,
		Real4* sortedRhoPreMu, uint* gridMarkerIndex, uint* cellStart,
		uint* cellEnd, uint numAllMarkers) {
	uint index = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;
	if (index >= numAllMarkers)
		return;

	// read particle data from sorted arrays
	Real3 posRadA = FETCH(sortedPosRad, index);
	Real4 rhoPreMuA = FETCH(sortedRhoPreMu, index);

	if (rhoPreMuA.w > -.1)
		return;

	// get address in grid
	int3 gridPos = calcGridPos(posRadA);

	Real densityShare = 0.0f;
	Real denominator = 0.0f;
	// examine neighbouring cells
	for (int z = -1; z <= 1; z++) {
		for (int y = -1; y <= 1; y++) {
			for (int x = -1; x <= 1; x++) {
				int3 neighbourPos = gridPos + mI3(x, y, z);
				collideCellDensityReInit(densityShare, denominator,
						neighbourPos, index, posRadA, sortedPosRad,
						sortedVelMas, sortedRhoPreMu, cellStart, cellEnd);
			}
		}
	}
	// write new velocity back to original unsorted location

	Real newDensity = densityShare + paramsD.markerMass * W3(0); //?$ include the particle in its summation as well
	Real newDenominator = denominator
			+ paramsD.markerMass * W3(0) / rhoPreMuA.x;
	if (rhoPreMuA.w < 0) {
		//		rhoPreMuA.x = newDensity; // old version
		rhoPreMuA.x = newDensity / newDenominator;  // correct version
	}
	rhoPreMuA.y = Eos(rhoPreMuA.x, rhoPreMuA.w);
	dummySortedRhoPreMu[index] = rhoPreMuA;
}
//--------------------------------------------------------------------------------------------------------------------------------
// without normalization
__global__ void ProjectDensityPressureToBCandBCE_D(Real4* dummySortedRhoPreMu,
		Real3* sortedPosRad, Real4* sortedRhoPreMu, uint* gridMarkerIndex,
		uint* cellStart, uint* cellEnd, uint numAllMarkers) {
	uint index = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;
	if (index >= numAllMarkers)
		return;

	// read particle data from sorted arrays
	Real3 posRadA = FETCH(sortedPosRad, index);
	Real4 rhoPreMuA = FETCH(sortedRhoPreMu, index);

	if (rhoPreMuA.w < -.1)
		return;

	// get address in grid
	int3 gridPos = calcGridPos(posRadA);

	Real3 distRhoPress =
	mR3((RESOLUTION_LENGTH_MULT + 2) * paramsD.HSML, rhoPreMuA.x, rhoPreMuA.y); //(large distance, rhoA, pA)
	// examine neighbouring cells
	for (int z = -1; z <= 1; z++) {
		for (int y = -1; y <= 1; y++) {
			for (int x = -1; x <= 1; x++) {
				int3 neighbourPos = gridPos + mI3(x, y, z);
				projectTheClosestFluidMarker(distRhoPress, neighbourPos, index,
						posRadA, sortedPosRad, sortedRhoPreMu, cellStart,
						cellEnd);
			}
		}
	}
	// write new velocity back to original unsorted location
	rhoPreMuA.x = distRhoPress.y;
	rhoPreMuA.y = distRhoPress.z;
	dummySortedRhoPreMu[index] = rhoPreMuA;
}
//--------------------------------------------------------------------------------------------------------------------------------
// without normalization
__global__ void CalcCartesianDataD(Real4* rho_Pres_CartD,
		Real4* vel_VelMag_CartD, Real3* sortedPosRad, Real3* sortedVelMas,
		Real4* sortedRhoPreMu, uint* gridMarkerIndex, uint* cellStart,
		uint* cellEnd, int3 cartesianGridDims, Real resolution) {
	uint index = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;
	if (index
			>= cartesianGridDims.x * cartesianGridDims.y * cartesianGridDims.z)
		return;

	int3 gridLoc;
	gridLoc.z = index / (cartesianGridDims.x * cartesianGridDims.y);
	gridLoc.y = (index % (cartesianGridDims.x * cartesianGridDims.y))
			/ cartesianGridDims.x;
	gridLoc.x = (index % (cartesianGridDims.x * cartesianGridDims.y))
			% cartesianGridDims.x;
	// alias cartesianGridDims = Dim,  you can say:   "index = (Dim.x * Dim.y) * gridLoc.z + Dim.x * gridLoc.y +
	// gridLoc.x"

	// get address in grid
	Real3 gridNodePos3 = mR3(gridLoc) * resolution + paramsD.worldOrigin;
	int3 gridPos = calcGridPos(gridNodePos3);

	Real3 vel_share = mR3(0.0f);
	Real4 rho_pres_share = mR4(0.0f);
	// examine neighbouring cells
	for (int z = -1; z <= 1; z++) {
		for (int y = -1; y <= 1; y++) {
			for (int x = -1; x <= 1; x++) {
				int3 neighbourPos = gridPos + mI3(x, y, z);
				calcOnCartesianShare(vel_share, rho_pres_share, neighbourPos,
				gridNodePos3, sortedPosRad, sortedVelMas, sortedRhoPreMu,
						cellStart, cellEnd);
			}
		}
	}
	// write new velocity back to original unsorted location
	//  uint originalIndex = gridMarkerIndex[index];

	// Real newDensity = densityShare + paramsD.markerMass * W3(0); //?$ include the particle in its summation as well
	// if (rhoPreMuA.w < -.1) { rhoPreMuA.x = newDensity; }
	// rhoPreMuA.y = Eos(rhoPreMuA.x, rhoPreMuA.w);
	//   oldRhoPreMu[originalIndex] = rhoPreMuA;
	/////printf("density %f\n", rhoPreMuA.x);
	/////printf("densityshare %f\n", densityShare);
	/////printf("gridPos x y z %d %d %d %f\n", gridPos.x, gridPos.y, gridPos.z, densityShare);
	rho_Pres_CartD[index] = rho_pres_share;
	vel_VelMag_CartD[index] = mR4(vel_share, length(vel_share));
}

//%%%%%%%%%%%%%%%%%%%%%%%%
//--------------------------------------------------------------------------------------------------------------------------------
//--------------------------------------------------------------------------------------------------------------------------------
// updates the fluid particles' properties, i.e. velocity, density, pressure, position
__global__ void UpdateFluidD(Real3* posRadD, Real3* velMasD, Real3* vel_XSPH_D,
		Real4* rhoPresMuD, Real4* derivVelRhoD, int2 updatePortion, Real dT, volatile bool *isErrorD) {
	uint index = blockIdx.x * blockDim.x + threadIdx.x;
	index += updatePortion.x; // updatePortion = [start, end] index of the update portion
	if (index >= updatePortion.y) {
		return;
	}
	Real4 derivVelRho = derivVelRhoD[index];
	Real4 rhoPresMu = rhoPresMuD[index];

	if (rhoPresMu.w < 0) {
		//-------------
		// ** position
		//-------------

		Real3 vel_XSPH = vel_XSPH_D[index];
		// 0** if you have rigid BCE, make sure to apply same tweaks to them, to satify action/reaction. Or apply tweak to
		// force in advance
		// 1*** let's tweak a little bit :)
		if (!(isfinite(vel_XSPH.x) && isfinite(vel_XSPH.y) && isfinite(vel_XSPH.z))) {
			if (paramsD.enableAggressiveTweak) {
				vel_XSPH = mR3(0);
			} else {
				printf("Error! particle vel_XSPH is NAN: thrown from SDKCollisionSystem.cu, UpdateFluidDKerner !\n");
				*isErrorD = true;
				return;
			}
		}
		if (length(vel_XSPH) > paramsD.tweakMultV * paramsD.HSML / paramsD.dT
				&& paramsD.enableTweak) {
			vel_XSPH *= (paramsD.tweakMultV * paramsD.HSML / paramsD.dT)
					/ length(vel_XSPH);
		}
		// 1*** end tweak

		Real3 posRad = posRadD[index];
		Real3 updatedPositon = posRad + vel_XSPH * dT;
		if (!(isfinite(updatedPositon.x) && isfinite(updatedPositon.y) && isfinite(updatedPositon.z))) {
			printf("Error! particle position is NAN: thrown from SDKCollisionSystem.cu, UpdateFluidDKernel !\n");
			*isErrorD = true;
			return;
		}
		posRadD[index] = updatedPositon;  // posRadD updated

		//-------------
		// ** velocity
		//-------------

		Real3 velMas = velMasD[index];
		Real3 updatedVelocity = velMas + mR3(derivVelRho) * dT;



		if (!(isfinite(updatedVelocity.x) && isfinite(updatedVelocity.y) && isfinite(updatedVelocity.z))) {
			if (paramsD.enableAggressiveTweak) {
				updatedVelocity = mR3(0);
			} else {
				printf("Error! particle updatedVelocity is NAN: thrown from SDKCollisionSystem.cu, UpdateFluidDKernel !\n");
				*isErrorD = true;
				return;
			}
		}
		// 2*** let's tweak a little bit :)
		if (length(updatedVelocity)
				> paramsD.tweakMultV * paramsD.HSML / paramsD.dT
				&& paramsD.enableTweak) {
			updatedVelocity *= (paramsD.tweakMultV * paramsD.HSML / paramsD.dT)
					/ length(updatedVelocity);
		}
		// 2*** end tweak

		velMasD[index] = updatedVelocity;

	}
	// 3*** let's tweak a little bit :)
	if (!(isfinite(derivVelRho.w))) {
		if (paramsD.enableAggressiveTweak) {
			derivVelRho.w = 0;
		} else {
			printf("Error! particle derivVelRho.w is NAN: thrown from SDKCollisionSystem.cu, UpdateFluidDKernel !\n");
			*isErrorD = true;
			return;
		}
	}
	if (fabs(derivVelRho.w) > paramsD.tweakMultRho * paramsD.rho0 / paramsD.dT
			&& paramsD.enableTweak) {
		derivVelRho.w *= (paramsD.tweakMultRho * paramsD.rho0 / paramsD.dT)
				/ fabs(derivVelRho.w);  // to take care of the sign as well
	}
	// 2*** end tweak
	Real rho2 = rhoPresMu.x + derivVelRho.w * dT; // rho update. (i.e. rhoPresMu.x), still not wriiten to global matrix
	rhoPresMu.y = Eos(rho2, rhoPresMu.w);
	rhoPresMu.x = rho2;
	if (!(isfinite(rhoPresMu.x) && isfinite(rhoPresMu.y) && isfinite(rhoPresMu.z) && isfinite(rhoPresMu.w))) {
		printf("Error! particle rho pressure is NAN: thrown from SDKCollisionSystem.cu, UpdateFluidDKernel !\n");
		*isErrorD = true;
		return;
	}
	rhoPresMuD[index] = rhoPresMu;  // rhoPresMuD updated
}

/**
 * @brief Copies the sortedVelXSPH to velXSPH according to indexing
 * @details [long description]
 *
 * @param vel_XSPH_D
 * @param vel_XSPH_Sorted_D Pointer to new sorted vel_XSPH vector
 * @param m_dGridMarkerIndex List of indeces used to sort vel_XSPH_D
 */

__global__ void CopySorted_vXSPH_dVdRho_to_original_kernel(Real3* vel_XSPH_D,
		Real4* derivVelRhoD,
		Real3* vel_XSPH_Sorted_D, Real4* sortedDerivVelRho_fsi_D,
		uint* mapOriginalToSorted) {
	uint index = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;
	if (index >= numObjectsD.numAllMarkers)
		return;
	vel_XSPH_D[index] = vel_XSPH_Sorted_D[mapOriginalToSorted[index]];
	derivVelRhoD[index] = sortedDerivVelRho_fsi_D[mapOriginalToSorted[index]];
}

//--------------------------------------------------------------------------------------------------------------------------------
// updates the fluid particles' properties, i.e. velocity, density, pressure, position
__global__ void UpdateKernelBoundary(
		Real4* rhoPresMuD, Real4* derivVelRhoD, int2 updatePortion, Real dT) {
	uint index = blockIdx.x * blockDim.x + threadIdx.x;
	index += updatePortion.x; // updatePortion = [start, end] index of the update portion
	if (index >= updatePortion.y) {
		return;
	}

	Real4 derivVelRho = derivVelRhoD[index];
	Real4 rhoPresMu = rhoPresMuD[index];
	Real rho2 = rhoPresMu.x + derivVelRho.w * dT; // rho update. (i.e. rhoPresMu.x), still not wriiten to global matrix
	rhoPresMu.y = Eos(rho2, rhoPresMu.w);
	rhoPresMu.x = rho2;
	if (!(isfinite(rhoPresMu.x) && isfinite(rhoPresMu.y) && isfinite(rhoPresMu.z) && isfinite(rhoPresMu.w))) {
							printf("Error! particle rp is NAN: thrown from SDKCollisionSystem.cu, UpdateKernelBoundary !\n");
				}
	rhoPresMuD[index] = rhoPresMu;  // rhoPresMuD updated
}

//--------------------------------------------------------------------------------------------------------------------------------
// applies periodic BC along x
__global__ void ApplyPeriodicBoundaryXKernel(Real3* posRadD,
		Real4* rhoPresMuD) {
	uint index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= numObjectsD.numAllMarkers) {
		return;
	}
	Real4 rhoPresMu = rhoPresMuD[index];
	if (fabs(rhoPresMu.w) < .1) {
		return;
	}  // no need to do anything if it is a boundary particle
	Real3 posRad = posRadD[index];
	if (posRad.x > paramsD.cMax.x) {
		posRad.x -= (paramsD.cMax.x - paramsD.cMin.x);
		posRadD[index] = posRad;
		if (rhoPresMu.w < -.1) {
			rhoPresMu.y = rhoPresMu.y + paramsD.deltaPress.x;
			rhoPresMuD[index] = rhoPresMu;
		}
		return;
	}
	if (posRad.x < paramsD.cMin.x) {
		posRad.x += (paramsD.cMax.x - paramsD.cMin.x);
		posRadD[index] = posRad;
		if (rhoPresMu.w < -.1) {
			rhoPresMu.y = rhoPresMu.y - paramsD.deltaPress.x;
			rhoPresMuD[index] = rhoPresMu;
		}
		return;
	}
}
//--------------------------------------------------------------------------------------------------------------------------------
// applies periodic BC along y
__global__ void ApplyPeriodicBoundaryYKernel(Real3* posRadD,
		Real4* rhoPresMuD) {
	uint index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= numObjectsD.numAllMarkers) {
		return;
	}
	Real4 rhoPresMu = rhoPresMuD[index];
	if (fabs(rhoPresMu.w) < .1) {
		return;
	}  // no need to do anything if it is a boundary particle
	Real3 posRad = posRadD[index];
	if (posRad.y > paramsD.cMax.y) {
		posRad.y -= (paramsD.cMax.y - paramsD.cMin.y);
		posRadD[index] = posRad;
		if (rhoPresMu.w < -.1) {
			rhoPresMu.y = rhoPresMu.y + paramsD.deltaPress.y;
			rhoPresMuD[index] = rhoPresMu;
		}
		return;
	}
	if (posRad.y < paramsD.cMin.y) {
		posRad.y += (paramsD.cMax.y - paramsD.cMin.y);
		posRadD[index] = posRad;
		if (rhoPresMu.w < -.1) {
			rhoPresMu.y = rhoPresMu.y - paramsD.deltaPress.y;
			rhoPresMuD[index] = rhoPresMu;
		}
		return;
	}
}
//--------------------------------------------------------------------------------------------------------------------------------
// applies periodic BC along z
__global__ void ApplyPeriodicBoundaryZKernel(Real3* posRadD,
		Real4* rhoPresMuD) {
	uint index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= numObjectsD.numAllMarkers) {
		return;
	}
	Real4 rhoPresMu = rhoPresMuD[index];
	if (fabs(rhoPresMu.w) < .1) {
		return;
	}  // no need to do anything if it is a boundary particle
	Real3 posRad = posRadD[index];
	if (posRad.z > paramsD.cMax.z) {
		posRad.z -= (paramsD.cMax.z - paramsD.cMin.z);
		posRadD[index] = posRad;
		if (rhoPresMu.w < -.1) {
			rhoPresMu.y = rhoPresMu.y + paramsD.deltaPress.z;
			rhoPresMuD[index] = rhoPresMu;
		}
		return;
	}
	if (posRad.z < paramsD.cMin.z) {
		posRad.z += (paramsD.cMax.z - paramsD.cMin.z);
		posRadD[index] = posRad;
		if (rhoPresMu.w < -.1) {
			rhoPresMu.y = rhoPresMu.y - paramsD.deltaPress.z;
			rhoPresMuD[index] = rhoPresMu;
		}
		return;
	}
}

//%%%%%%%%%%%%%%%%%%%%%%%%
//--------------------------------------------------------------------------------------------------------------------------------
void allocateArray(void** devPtr, size_t size) {
	hipMalloc(devPtr, size);
}
//--------------------------------------------------------------------------------------------------------------------------------
void freeArray(void* devPtr) {
	hipFree(devPtr);
}

/**
 * @brief iDivUp
 * @details Round a / b to nearest higher integer value
 *
 * @param a numerator
 * @param b denominator
 *
 * @return ceil(a/b)
 */
uint iDivUp(uint a, uint b) {
	return (a % b != 0) ? (a / b + 1) : (a / b);
}

/**
 * @brief computeGridSize
 * @details Compute grid and thread block size for a given number of elements
 *
 * @param n Total number of elements. Each elements needs a thread to be computed
 * @param blockSize Number of threads per block.
 * @param numBlocks output
 * @param numThreads Output: number of threads per block
 */
void computeGridSize(uint n, uint blockSize, uint& numBlocks,
		uint& numThreads) {
	uint n2 = (n == 0) ? 1 : n;
	numThreads = min(blockSize, n2);
	numBlocks = iDivUp(n2, numThreads);
}

/**
 * @brief [brief description]
 * @details [long description]
 *
 * @param hostParams [description]
 * @param numObjects [description]
 */
void setParameters(SimParams* hostParams, NumberOfObjects* numObjects) {
	// copy parameters to constant memory
	hipMemcpyToSymbolAsync(HIP_SYMBOL(paramsD), hostParams, sizeof(SimParams));
	hipMemcpyToSymbolAsync(HIP_SYMBOL(numObjectsD), numObjects, sizeof(NumberOfObjects));
}

/**
 * @brief Wrapper function for calcHashD
 * @details See SDKCollisionSystem.cuh for more info
 */
void calcHash(thrust::device_vector<uint>& gridMarkerHash,
		thrust::device_vector<uint>& gridMarkerIndex,
		thrust::device_vector<Real3>& posRad,
		int numAllMarkers) {


	bool *isErrorH, *isErrorD;
	isErrorH = (bool *)malloc(sizeof(bool));
	hipMalloc((void**) &isErrorD, sizeof(bool));
	*isErrorH = false;
	hipMemcpy(isErrorD, isErrorH, sizeof(bool), hipMemcpyHostToDevice);
	//------------------------------------------------------------------------
	/* Is there a need to optimize the number of threads used at once? */
	uint numThreads, numBlocks;
	computeGridSize(numAllMarkers, 256, numBlocks, numThreads);
	/* Execute Kernel */
	calcHashD<<<numBlocks, numThreads>>>(U1CAST(gridMarkerHash),
			U1CAST(gridMarkerIndex), mR3CAST(posRad),
			numAllMarkers, isErrorD);

	/* Check for errors in kernel execution */
	hipDeviceSynchronize();
	cudaCheckError();
	//------------------------------------------------------------------------
	hipMemcpy(isErrorH, isErrorD, sizeof(bool), hipMemcpyDeviceToHost);
	if (*isErrorH == true) {
		throw std::runtime_error ("Error! program crashed in  calcHashD!\n");
	}
	hipFree(isErrorD);
	free(isErrorH);
}

/**
 * @brief Wrapper function for reorderDataAndFindCellStartD
 * @details
 * 		See SDKCollisionSystem.cuh for brief.
 */
void reorderDataAndFindCellStart(thrust::device_vector<uint>& cellStart,
		thrust::device_vector<uint>& cellEnd,
		thrust::device_vector<Real3>& sortedPosRad,
		thrust::device_vector<Real3>& sortedVelMas,
		thrust::device_vector<Real4>& sortedRhoPreMu,

		thrust::device_vector<uint>& gridMarkerHash,
		thrust::device_vector<uint>& gridMarkerIndex,

		thrust::device_vector<uint>& mapOriginalToSorted,

		thrust::device_vector<Real3>& oldPosRad,
		thrust::device_vector<Real3>& oldVelMas,
		thrust::device_vector<Real4>& oldRhoPreMu, uint numAllMarkers,
		uint numCells) {
	uint numThreads, numBlocks;
	computeGridSize(numAllMarkers, 256, numBlocks, numThreads); //?$ 256 is blockSize

	/* Set all cells to empty */
//	hipMemset(U1CAST(cellStart), 0xffffffff, numCells * sizeof(uint));
	thrust::fill(cellStart.begin(), cellStart.end(), 0);
	thrust::fill(cellEnd.begin(), cellEnd.end(), 0);

	//#if USE_TEX
	//#if 0
	//    cutilSafeCall(hipBindTexture(0, oldPosTex, oldPosRad, numAllMarkers*sizeof(Real4)));
	//    cutilSafeCall(hipBindTexture(0, oldVelTex, oldVelMas, numAllMarkers*sizeof(Real4)));
	//#endif

	uint smemSize = sizeof(uint) * (numThreads + 1);
	reorderDataAndFindCellStartD<<<numBlocks, numThreads, smemSize>>>(
			U1CAST(cellStart), U1CAST(cellEnd), mR3CAST(sortedPosRad),
			mR3CAST(sortedVelMas), mR4CAST(sortedRhoPreMu),
			U1CAST(gridMarkerHash), U1CAST(gridMarkerIndex),
			U1CAST(mapOriginalToSorted), mR3CAST(oldPosRad), mR3CAST(oldVelMas),
			mR4CAST(oldRhoPreMu), numAllMarkers);
	hipDeviceSynchronize();
	cudaCheckError()
	;

	// unroll sorted index to have the location of original particles in the sorted arrays
	thrust::device_vector<uint> dummyIndex = gridMarkerIndex;
	thrust::sort_by_key(dummyIndex.begin(), dummyIndex.end(),
			mapOriginalToSorted.begin());
	dummyIndex.clear();
	//#if USE_TEX
	//#if 0
	//    cutilSafeCall(hipUnbindTexture(oldPosTex));
	//    cutilSafeCall(hipUnbindTexture(oldVelTex));
	//#endif
}

/**
 * @brief Wrapper function for newVel_XSPH_D
 */
void RecalcVelocity_XSPH(thrust::device_vector<Real3>& vel_XSPH_Sorted_D,
		thrust::device_vector<Real3>& sortedPosRad,
		thrust::device_vector<Real3>& sortedVelMas,
		thrust::device_vector<Real4>& sortedRhoPreMu,
		thrust::device_vector<uint>& gridMarkerIndex,
		thrust::device_vector<uint>& cellStart,
		thrust::device_vector<uint>& cellEnd, uint numAllMarkers,
		uint numCells) {

	bool *isErrorH, *isErrorD;
	isErrorH = (bool *)malloc(sizeof(bool));
	hipMalloc((void**) &isErrorD, sizeof(bool));
	*isErrorH = false;
	hipMemcpy(isErrorD, isErrorH, sizeof(bool), hipMemcpyHostToDevice);
	//------------------------------------------------------------------------
	/* thread per particle */
	uint numThreads, numBlocks;
	computeGridSize(numAllMarkers, 64, numBlocks, numThreads);

	/* Execute the kernel */
	newVel_XSPH_D<<<numBlocks, numThreads>>>(mR3CAST(vel_XSPH_Sorted_D),
			mR3CAST(sortedPosRad), mR3CAST(sortedVelMas),
			mR4CAST(sortedRhoPreMu), U1CAST(gridMarkerIndex), U1CAST(cellStart),
			U1CAST(cellEnd), numAllMarkers, isErrorD);

	hipDeviceSynchronize();
	cudaCheckError();
	//------------------------------------------------------------------------
	hipMemcpy(isErrorH, isErrorD, sizeof(bool), hipMemcpyDeviceToHost);
	if (*isErrorH == true) {
		throw std::runtime_error ("Error! program crashed in  newVel_XSPH_D!\n");
	}
	hipFree(isErrorD);
	free(isErrorH);
}
//--------------------------------------------------------------------------------------------------------------------------------
void RecalcSortedVelocityPressure_BCE(
		thrust::device_vector<Real3>& velMas_ModifiedBCE,
		thrust::device_vector<Real4>& rhoPreMu_ModifiedBCE,
		const thrust::device_vector<Real3>& sortedPosRad,
		const thrust::device_vector<Real3>& sortedVelMas,
		const thrust::device_vector<Real4>& sortedRhoPreMu,
		const thrust::device_vector<uint>& cellStart,
		const thrust::device_vector<uint>& cellEnd,
		const thrust::device_vector<uint>& mapOriginalToSorted,
		const thrust::device_vector<Real3>& bceAcc,
		int2 updatePortion) {

	bool *isErrorH, *isErrorD;
	isErrorH = (bool *)malloc(sizeof(bool));
	hipMalloc((void**) &isErrorD, sizeof(bool));
	*isErrorH = false;
	hipMemcpy(isErrorD, isErrorH, sizeof(bool), hipMemcpyHostToDevice);
	//------------------------------------------------------------------------

	// thread per particle
	uint numThreads, numBlocks;
	computeGridSize(updatePortion.y - updatePortion.x, 64, numBlocks, numThreads);

	new_BCE_VelocityPressure<<<numBlocks, numThreads>>>(
			mR3CAST(velMas_ModifiedBCE),
			mR4CAST(rhoPreMu_ModifiedBCE),  // input: sorted velocities
			mR3CAST(sortedPosRad), mR3CAST(sortedVelMas),
			mR4CAST(sortedRhoPreMu), U1CAST(cellStart), U1CAST(cellEnd), U1CAST(mapOriginalToSorted),
			mR3CAST(bceAcc),
			updatePortion,
			isErrorD);

	hipDeviceSynchronize();
	cudaCheckError()

	//------------------------------------------------------------------------
	hipMemcpy(isErrorH, isErrorD, sizeof(bool), hipMemcpyDeviceToHost);
	if (*isErrorH == true) {
		throw std::runtime_error ("Error! program crashed in  new_BCE_VelocityPressure!\n");
	}
	hipFree(isErrorD);
	free(isErrorH);
}

//--------------------------------------------------------------------------------------------------------------------------------
void CalcBCE_Stresses(thrust::device_vector<Real3>& devStressD,
		thrust::device_vector<Real3>& volStressD,
		thrust::device_vector<Real4>& mainStressD,
		thrust::device_vector<Real3>& sortedPosRad,
		thrust::device_vector<Real3>& sortedVelMas,
		thrust::device_vector<Real4>& sortedRhoPreMu,
		thrust::device_vector<uint>& mapOriginalToSorted,
		thrust::device_vector<uint>& cellStart,
		thrust::device_vector<uint>& cellEnd, int numBCE) {
	// thread per particle
	uint numThreads, numBlocks;
	computeGridSize(numBCE, 128, numBlocks, numThreads);
	CalcBCE_Stresses_kernel<<<numBlocks, numThreads>>>(mR3CAST(devStressD),
			mR3CAST(volStressD), mR3CAST(sortedPosRad), mR3CAST(sortedVelMas),
			mR4CAST(sortedRhoPreMu), U1CAST(mapOriginalToSorted),
			U1CAST(cellStart), U1CAST(cellEnd), numBCE);

	hipDeviceSynchronize();
	cudaCheckError()
	;

	CalcBCE_MainStresses_kernel<<<numBlocks, numThreads>>>(mR4CAST(mainStressD),
			mR3CAST(devStressD), mR3CAST(volStressD), numBCE);

	hipDeviceSynchronize();
	cudaCheckError()
	;
}

/**
 * @brief Wrapper function for collide
 * @details
 * 		See SDKCollisionSystem.cuh for informaton on collide
 */
void collide(thrust::device_vector<Real4>& sortedDerivVelRho_fsi_D,
		thrust::device_vector<Real3>& sortedPosRad,
		thrust::device_vector<Real3>& sortedVelMas,
		thrust::device_vector<Real3>& vel_XSPH_Sorted_D,
		thrust::device_vector<Real4>& sortedRhoPreMu,
		thrust::device_vector<Real3>& velMas_ModifiedBCE,
		thrust::device_vector<Real4>& rhoPreMu_ModifiedBCE,

		thrust::device_vector<uint>& gridMarkerIndex,
		thrust::device_vector<uint>& cellStart,
		thrust::device_vector<uint>& cellEnd, uint numAllMarkers, uint numCells,
		Real dT) {
	//#if USE_TEX
	//    cutilSafeCall(hipBindTexture(0, oldPosTex, sortedPosRad, numAllMarkers*sizeof(Real4)));
	//    cutilSafeCall(hipBindTexture(0, oldVelTex, sortedVelMas, numAllMarkers*sizeof(Real4)));
	//    cutilSafeCall(hipBindTexture(0, cellStartTex, cellStart, numCells*sizeof(uint)));
	//    cutilSafeCall(hipBindTexture(0, cellEndTex, cellEnd, numCells*sizeof(uint)));
	//#endif

	bool *isErrorH, *isErrorD;
	isErrorH = (bool *)malloc(sizeof(bool));
	hipMalloc((void**) &isErrorD, sizeof(bool));
	*isErrorH = false;
	hipMemcpy(isErrorD, isErrorH, sizeof(bool), hipMemcpyHostToDevice);
	//------------------------------------------------------------------------
	// thread per particle
	uint numThreads, numBlocks;
	computeGridSize(numAllMarkers, 64, numBlocks, numThreads);

	// execute the kernel
	collideD<<<numBlocks, numThreads>>>(mR4CAST(sortedDerivVelRho_fsi_D),
			mR3CAST(sortedPosRad), mR3CAST(sortedVelMas),
			mR3CAST(vel_XSPH_Sorted_D), mR4CAST(sortedRhoPreMu),
			mR3CAST(velMas_ModifiedBCE), mR4CAST(rhoPreMu_ModifiedBCE), U1CAST(gridMarkerIndex),
			U1CAST(cellStart), U1CAST(cellEnd),
			numAllMarkers, isErrorD);

	hipDeviceSynchronize();
	cudaCheckError();
	//------------------------------------------------------------------------
	hipMemcpy(isErrorH, isErrorD, sizeof(bool), hipMemcpyDeviceToHost);
	if (*isErrorH == true) {
		throw std::runtime_error ("Error! program crashed in  collideD!\n");
	}
	hipFree(isErrorD);
	free(isErrorH);


//					// unroll sorted index to have the location of original particles in the sorted arrays
//					thrust::device_vector<uint> dummyIndex = gridMarkerIndex;
//					thrust::sort_by_key(dummyIndex.begin(), dummyIndex.end(),
//							derivVelRhoD.begin());
//					dummyIndex.clear();


	//#if USE_TEX
	//    cutilSafeCall(hipUnbindTexture(oldPosTex));
	//    cutilSafeCall(hipUnbindTexture(oldVelTex));
	//    cutilSafeCall(hipUnbindTexture(cellStartTex));
	//    cutilSafeCall(hipUnbindTexture(cellEndTex));
	//#endif
}
//--------------------------------------------------------------------------------------------------------------------------------
void ReCalcDensity(thrust::device_vector<Real4>& oldRhoPreMu,
		thrust::device_vector<Real3>& sortedPosRad,
		thrust::device_vector<Real3>& sortedVelMas,
		thrust::device_vector<Real4>& sortedRhoPreMu,
		thrust::device_vector<uint>& gridMarkerIndex,
		thrust::device_vector<uint>& cellStart,
		thrust::device_vector<uint>& cellEnd, uint numAllMarkers) {
	//#if USE_TEX
	//    cutilSafeCall(hipBindTexture(0, oldPosTex, sortedPosRad, numAllMarkers*sizeof(Real4)));
	//    cutilSafeCall(hipBindTexture(0, oldVelTex, sortedVelMas, numAllMarkers*sizeof(Real4)));
	//    cutilSafeCall(hipBindTexture(0, cellStartTex, cellStart, numCells*sizeof(uint)));
	//    cutilSafeCall(hipBindTexture(0, cellEndTex, cellEnd, numCells*sizeof(uint)));
	//#endif

	// thread per particle
	uint numThreads, numBlocks;
	computeGridSize(numAllMarkers, 64, numBlocks, numThreads);

	// execute the kernel
	thrust::device_vector<Real4> dummySortedRhoPreMu = sortedRhoPreMu;
	ReCalcDensityD_F1<<<numBlocks, numThreads>>>(mR4CAST(dummySortedRhoPreMu), mR3CAST(sortedPosRad),
			mR3CAST(sortedVelMas), mR4CAST(sortedRhoPreMu),
			U1CAST(gridMarkerIndex), U1CAST(cellStart), U1CAST(cellEnd),
			numAllMarkers);

	hipDeviceSynchronize();
	cudaCheckError();
	CopySortedToOriginal_Invasive_R4(oldRhoPreMu, dummySortedRhoPreMu, gridMarkerIndex);
	dummySortedRhoPreMu.clear();

	//#if USE_TEX
	//    cutilSafeCall(hipUnbindTexture(oldPosTex));
	//    cutilSafeCall(hipUnbindTexture(oldVelTex));
	//    cutilSafeCall(hipUnbindTexture(cellStartTex));
	//    cutilSafeCall(hipUnbindTexture(cellEndTex));
	//#endif
}
//--------------------------------------------------------------------------------------------------------------------------------
void ProjectDensityPressureToBCandBCE(thrust::device_vector<Real4>& oldRhoPreMu,
		thrust::device_vector<Real3>& sortedPosRad,
		thrust::device_vector<Real4>& sortedRhoPreMu,
		thrust::device_vector<uint>& gridMarkerIndex,
		thrust::device_vector<uint>& cellStart,
		thrust::device_vector<uint>& cellEnd, uint numAllMarkers) {
	//#if USE_TEX
	//    cutilSafeCall(hipBindTexture(0, oldPosTex, sortedPosRad, numAllMarkers*sizeof(Real4)));
	//    cutilSafeCall(hipBindTexture(0, oldVelTex, sortedVelMas, numAllMarkers*sizeof(Real4)));
	//    cutilSafeCall(hipBindTexture(0, cellStartTex, cellStart, numCells*sizeof(uint)));
	//    cutilSafeCall(hipBindTexture(0, cellEndTex, cellEnd, numCells*sizeof(uint)));
	//#endif

	// thread per particle
	uint numThreads, numBlocks;
	computeGridSize(numAllMarkers, 64, numBlocks, numThreads);

	// execute the kernel
	thrust::device_vector<Real4>& dummySortedRhoPreMu = sortedRhoPreMu;
	ProjectDensityPressureToBCandBCE_D<<<numBlocks, numThreads>>>(
			mR4CAST(dummySortedRhoPreMu), mR3CAST(sortedPosRad),
			mR4CAST(sortedRhoPreMu), U1CAST(gridMarkerIndex), U1CAST(cellStart),
			U1CAST(cellEnd), numAllMarkers);

	hipDeviceSynchronize();
	cudaCheckError();
	CopySortedToOriginal_Invasive_R4(oldRhoPreMu, dummySortedRhoPreMu, gridMarkerIndex);
	dummySortedRhoPreMu.clear();

	//#if USE_TEX
	//    cutilSafeCall(hipUnbindTexture(oldPosTex));
	//    cutilSafeCall(hipUnbindTexture(oldVelTex));
	//    cutilSafeCall(hipUnbindTexture(cellStartTex));
	//    cutilSafeCall(hipUnbindTexture(cellEndTex));
	//#endif
}
//--------------------------------------------------------------------------------------------------------------------------------
void CalcCartesianData(thrust::device_vector<Real4>& rho_Pres_CartD,
		thrust::device_vector<Real4>& vel_VelMag_CartD,
		thrust::device_vector<Real3>& sortedPosRad,
		thrust::device_vector<Real3>& sortedVelMas,
		thrust::device_vector<Real4>& sortedRhoPreMu,
		thrust::device_vector<uint>& gridMarkerIndex,
		thrust::device_vector<uint>& cellStart,
		thrust::device_vector<uint>& cellEnd, uint cartesianGridSize,
		int3 cartesianGridDims, Real resolution) {

	// thread per particle
	uint numThreads, numBlocks;
	computeGridSize(cartesianGridSize, 64, numBlocks, numThreads);

	// execute the kernel
	CalcCartesianDataD<<<numBlocks, numThreads>>>(mR4CAST(rho_Pres_CartD),
			mR4CAST(vel_VelMag_CartD), mR3CAST(sortedPosRad),
			mR3CAST(sortedVelMas), mR4CAST(sortedRhoPreMu),
			U1CAST(gridMarkerIndex), U1CAST(cellStart), U1CAST(cellEnd),
			cartesianGridDims, resolution);

	hipDeviceSynchronize();
	cudaCheckError()
	;

	//#if USE_TEX
	//    cutilSafeCall(hipUnbindTexture(oldPosTex));
	//    cutilSafeCall(hipUnbindTexture(oldVelTex));
	//    cutilSafeCall(hipUnbindTexture(cellStartTex));
	//    cutilSafeCall(hipUnbindTexture(cellEndTex));
	//#endif
}

//%%%%%%%%%%%%%%%%%%%%%%%%%%
//--------------------------------------------------------------------------------------------------------------------------------
// updates the fluid particles by calling UpdateFluidD
void UpdateFluid(thrust::device_vector<Real3>& posRadD,
		thrust::device_vector<Real3>& velMasD,
		thrust::device_vector<Real3>& vel_XSPH_D,
		thrust::device_vector<Real4>& rhoPresMuD,
		thrust::device_vector<Real4>& derivVelRhoD,
		const thrust::host_vector<int4>& referenceArray, Real dT) {

//	int4 referencePortion = referenceArray[0];
//	if (referencePortion.z != -1) {
//		printf("error in UpdateFluid, accessing non fluid\n");
//		return;
//	}
//	int2 updatePortion = mI2(referencePortion);
	int2 updatePortion = mI2(0, referenceArray[referenceArray.size() - 1].y);
	// int2 updatePortion = mI2(referenceArray[0].x, referenceArray[0].y);

	bool *isErrorH, *isErrorD;
	isErrorH = (bool *)malloc(sizeof(bool));
	hipMalloc((void**) &isErrorD, sizeof(bool));
	*isErrorH = false;
	hipMemcpy(isErrorD, isErrorH, sizeof(bool), hipMemcpyHostToDevice);
	//------------------------
	uint nBlock_UpdateFluid, nThreads;
	computeGridSize(updatePortion.y - updatePortion.x, 128, nBlock_UpdateFluid,
			nThreads);
	UpdateFluidD<<<nBlock_UpdateFluid, nThreads>>>(mR3CAST(posRadD),
			mR3CAST(velMasD), mR3CAST(vel_XSPH_D), mR4CAST(rhoPresMuD),
			mR4CAST(derivVelRhoD), updatePortion, dT, isErrorD);
	hipDeviceSynchronize();
	cudaCheckError();
	//------------------------
	hipMemcpy(isErrorH, isErrorD, sizeof(bool), hipMemcpyDeviceToHost);
	if (*isErrorH == true) {
		throw std::runtime_error ("Error! program crashed in  UpdateFluidD!\n");
	}
	hipFree(isErrorD);
	free(isErrorH);
}

////--------------------------------------------------------------------------------------------------------------------------------
//void CopySorted_vXSPH_dVdRho_to_original(thrust::device_vector<Real3>& vel_XSPH_D,
//		thrust::device_vector<Real4>& derivVelRhoD,
//		thrust::device_vector<Real3>& vel_XSPH_Sorted_D,
//		thrust::device_vector<Real4>& sortedDerivVelRho_fsi_D,
//		thrust::device_vector<uint>& mapOriginalToSorted, int numAllMarkers) {
//	uint nBlock_NumSpheres, nThreads_SphMarkers;
//	computeGridSize(numAllMarkers, 256, nBlock_NumSpheres, nThreads_SphMarkers);
//	CopySorted_vXSPH_dVdRho_to_original_kernel<<<nBlock_NumSpheres, nThreads_SphMarkers>>>(
//			mR3CAST(vel_XSPH_D), mR4CAST(derivVelRhoD),
//			mR3CAST(vel_XSPH_Sorted_D),mR4CAST(sortedDerivVelRho_fsi_D),
//			U1CAST(mapOriginalToSorted));
//	hipDeviceSynchronize();
//	cudaCheckError()
//	;
//}
//--------------------------------------------------------------------------------------------------------------------------------
// use invasive to avoid one extra copy. However, keep in mind that sorted is changed.
void CopySortedToOriginal_Invasive_R3(thrust::device_vector<Real3>& original,
		thrust::device_vector<Real3>& sorted,
		const thrust::device_vector<uint>& gridMarkerIndex) {
	thrust::device_vector<uint> dummyMarkerIndex = gridMarkerIndex;
	thrust::sort_by_key(dummyMarkerIndex.begin(), dummyMarkerIndex.end(),
			sorted.begin());
	dummyMarkerIndex.clear();
	thrust::copy(sorted.begin(), sorted.end(), original.begin());
}
//--------------------------------------------------------------------------------------------------------------------------------
void CopySortedToOriginal_NonInvasive_R3(thrust::device_vector<Real3>& original,
		thrust::device_vector<Real3>& sorted,
		const thrust::device_vector<uint>& gridMarkerIndex) {
	thrust::device_vector<Real3> dummySorted = sorted;
	CopySortedToOriginal_Invasive_R3(original, dummySorted, gridMarkerIndex);
}
//--------------------------------------------------------------------------------------------------------------------------------
// use invasive to avoid one extra copy. However, keep in mind that sorted is changed.
void CopySortedToOriginal_Invasive_R4(thrust::device_vector<Real4>& original,
		thrust::device_vector<Real4>& sorted,
		const thrust::device_vector<uint>& gridMarkerIndex) {
	thrust::device_vector<uint> dummyMarkerIndex = gridMarkerIndex;
	thrust::sort_by_key(dummyMarkerIndex.begin(), dummyMarkerIndex.end(),
			sorted.begin());
	dummyMarkerIndex.clear();
	thrust::copy(sorted.begin(), sorted.end(), original.begin());
}
//--------------------------------------------------------------------------------------------------------------------------------
void CopySortedToOriginal_NonInvasive_R4(thrust::device_vector<Real4>& original,
		thrust::device_vector<Real4>& sorted,
		const thrust::device_vector<uint>& gridMarkerIndex) {
	thrust::device_vector<Real4> dummySorted = sorted;
	CopySortedToOriginal_Invasive_R4(original, dummySorted, gridMarkerIndex);
}

//--------------------------------------------------------------------------------------------------------------------------------
// updates the fluid particles by calling UpdateBoundary
void UpdateBoundary(
		thrust::device_vector<Real4>& rhoPresMuD,
		thrust::device_vector<Real4>& derivVelRhoD,
		const thrust::host_vector<int4>& referenceArray, Real dT) {
	int4 referencePortion = referenceArray[1];
	if (referencePortion.z != 0) {
		printf("error in UpdateBoundary, accessing non boundary\n");
		return;
	}
	int2 updatePortion = mI2(referencePortion);

	uint nBlock_UpdateFluid, nThreads;
	computeGridSize(updatePortion.y - updatePortion.x, 128, nBlock_UpdateFluid,
			nThreads);
	UpdateKernelBoundary<<<nBlock_UpdateFluid, nThreads>>>(mR4CAST(rhoPresMuD), mR4CAST(derivVelRhoD),
			updatePortion, dT);
	hipDeviceSynchronize();
	cudaCheckError()
	;
}

/**
 * @brief ApplyBoundarySPH_Markers
 * @details
 * 		See SDKCollisionSystem.cuh for more info
 */
void ApplyBoundarySPH_Markers(thrust::device_vector<Real3>& posRadD,
		thrust::device_vector<Real4>& rhoPresMuD, int numAllMarkers) {
	uint nBlock_NumSpheres, nThreads_SphMarkers;
	computeGridSize(numAllMarkers, 256, nBlock_NumSpheres, nThreads_SphMarkers);
	ApplyPeriodicBoundaryXKernel<<<nBlock_NumSpheres, nThreads_SphMarkers>>>(
			mR3CAST(posRadD), mR4CAST(rhoPresMuD));
	hipDeviceSynchronize();
	cudaCheckError()
	;
	// these are useful anyway for out of bound particles
	ApplyPeriodicBoundaryYKernel<<<nBlock_NumSpheres, nThreads_SphMarkers>>>(
			mR3CAST(posRadD), mR4CAST(rhoPresMuD));
	hipDeviceSynchronize();
	cudaCheckError()
	;
	ApplyPeriodicBoundaryZKernel<<<nBlock_NumSpheres, nThreads_SphMarkers>>>(
			mR3CAST(posRadD), mR4CAST(rhoPresMuD));
	hipDeviceSynchronize();
	cudaCheckError()
	;

	//	SetOutputPressureToZero_X<<<nBlock_NumSpheres, nThreads_SphMarkers>>>(mR3CAST(posRadD), mR4CAST(rhoPresMuD));
	//    hipDeviceSynchronize();
	//    cudaCheckError();
}
