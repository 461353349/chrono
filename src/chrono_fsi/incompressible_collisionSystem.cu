#include "hip/hip_runtime.h"
/*
 * SDKCollisionSystem.cu
 *
 *  Created on: Mar 2, 2013
 *      Author: Arman Pazouki, Milad Rakhsha
 */
#include <stdexcept>
#include <thrust/sort.h>
#include "chrono_fsi/incompressible_collisionSystem.cuh"


//#include "extraOptionalFunctions.cuh"
//#include "SDKCollisionSystemAdditional.cuh"

/**
 * @brief calcGridHash
 * @details  See SDKCollisionSystem.cuh
 */
__device__ int3 calcGridPos(Real3 p) {
	int3 gridPos;
	gridPos.x = floor((p.x - paramsD.worldOrigin.x) / paramsD.cellSize.x);
	gridPos.y = floor((p.y - paramsD.worldOrigin.y) / paramsD.cellSize.y);
	gridPos.z = floor((p.z - paramsD.worldOrigin.z) / paramsD.cellSize.z);
	return gridPos;
}

/**
 * @brief calcGridHash
 * @details  See SDKCollisionSystem.cuh
 */
__device__ uint calcGridHash(int3 gridPos) {
	gridPos.x -= ((gridPos.x >= paramsD.gridSize.x) ? paramsD.gridSize.x : 0);
	gridPos.y -= ((gridPos.y >= paramsD.gridSize.y) ? paramsD.gridSize.y : 0);
	gridPos.z -= ((gridPos.z >= paramsD.gridSize.z) ? paramsD.gridSize.z : 0);

	gridPos.x += ((gridPos.x < 0) ? paramsD.gridSize.x : 0);
	gridPos.y += ((gridPos.y < 0) ? paramsD.gridSize.y : 0);
	gridPos.z += ((gridPos.z < 0) ? paramsD.gridSize.z : 0);

	return __umul24(__umul24(gridPos.z, paramsD.gridSize.y), paramsD.gridSize.x)
			+ __umul24(gridPos.y, paramsD.gridSize.x) + gridPos.x;
}

/**
 * @brief calcGridHash
 * @details  See SDKCollisionSystem.cuh
 */
__device__ inline Real4 DifVelocityRho_implicit(Real3& dist3, Real& d, Real3 posRadA, Real3 posRadB,
		Real3& velMasA, Real3& velMasB,
		Real4& rhoPresMuA,
		Real4& rhoPresMuB, Real multViscosity) {
	// TODO
	// Milad: need to change this
}

//--------------------------------------------------------------------------------------------------------------------------------
// collide a particle against all other particles in a given cell
// Arman : revisit equation 10 of tech report, is it only on fluid or it is on all markers
__device__ void BCE_modification_Share(
		Real3& sumVW,
		Real& sumWAll,
		Real3& sumRhoRW,
		Real& sumPW,
		Real& sumWFluid,
		int& isAffectedV, int& isAffectedP, int3 gridPos,
		Real3 posRadA, Real3* sortedPosRad, Real3* sortedVelMas,
		Real4* sortedRhoPreMu, uint* cellStart, uint* cellEnd) {
	uint gridHash = calcGridHash(gridPos);
	// get start of bucket for this cell
	uint startIndex = FETCH(cellStart, gridHash);
	if (startIndex != 0xffffffff) {  // cell is not empty
		// iterate over particles in this cell
		uint endIndex = FETCH(cellEnd, gridHash);

		for (uint j = startIndex; j < endIndex; j++) {
			Real3 posRadB = FETCH(sortedPosRad, j);
			Real3 dist3 = Distance(posRadA, posRadB);
			Real d = length(dist3);
			Real4 rhoPresMuB = FETCH(sortedRhoPreMu, j);
			if (d > RESOLUTION_LENGTH_MULT * paramsD.HSML || rhoPresMuB.w > -.1)
				continue;

			Real Wd = W3(d);
			Real WdOvRho = Wd / rhoPresMuB.x;
			isAffectedV = 1;
			Real3 velMasB = FETCH(sortedVelMas, j);
			sumVW += velMasB * WdOvRho;
			sumWAll += WdOvRho;

			isAffectedP = 1;
			sumRhoRW += rhoPresMuB.x * dist3 * WdOvRho;
			sumPW += rhoPresMuB.y * WdOvRho;
			sumWFluid += WdOvRho;
		}
	}
}
//--------------------------------------------------------------------------------------------------------------------------------
// modify pressure for body force
__device__ __inline__ void modifyPressure(Real4& rhoPresMuB,
		const Real3& dist3Alpha) {
	// body force in x direction
	rhoPresMuB.y =
			(dist3Alpha.x > 0.5 * paramsD.boxDims.x) ?
					(rhoPresMuB.y - paramsD.deltaPress.x) : rhoPresMuB.y;
	rhoPresMuB.y =
			(dist3Alpha.x < -0.5 * paramsD.boxDims.x) ?
					(rhoPresMuB.y + paramsD.deltaPress.x) : rhoPresMuB.y;
	// body force in x direction
	rhoPresMuB.y =
			(dist3Alpha.y > 0.5 * paramsD.boxDims.y) ?
					(rhoPresMuB.y - paramsD.deltaPress.y) : rhoPresMuB.y;
	rhoPresMuB.y =
			(dist3Alpha.y < -0.5 * paramsD.boxDims.y) ?
					(rhoPresMuB.y + paramsD.deltaPress.y) : rhoPresMuB.y;
	// body force in x direction
	rhoPresMuB.y =
			(dist3Alpha.z > 0.5 * paramsD.boxDims.z) ?
					(rhoPresMuB.y - paramsD.deltaPress.z) : rhoPresMuB.y;
	rhoPresMuB.y =
			(dist3Alpha.z < -0.5 * paramsD.boxDims.z) ?
					(rhoPresMuB.y + paramsD.deltaPress.z) : rhoPresMuB.y;
}
//--------------------------------------------------------------------------------------------------------------------------------
// collide a particle against all other particles in a given cell
__device__ Real4 collideCell_implicit(int3 gridPos, uint index, Real3 posRadA,
		Real3 velMasA, Real4 rhoPresMuA, Real3* sortedPosRad,
		Real3* sortedVelMas, Real4* sortedRhoPreMu,
		Real3* velMas_ModifiedBCE, Real4* rhoPreMu_ModifiedBCE, uint* gridMarkerIndex,
		uint* cellStart, uint* cellEnd) {

	uint gridHash = calcGridHash(gridPos);
	// get start of bucket for this cell
	Real4 derivVelRho = mR4(0);

	uint startIndex = FETCH(cellStart, gridHash);
	if (startIndex == 0xffffffff) { // cell is not empty
		return derivVelRho;
	}
	// iterate over particles in this cell
	uint endIndex = FETCH(cellEnd, gridHash);

	for (uint j = startIndex; j < endIndex; j++) {
		if (j != index) {  // check not colliding with self
			Real3 posRadB = FETCH(sortedPosRad, j);
			Real3 dist3Alpha = posRadA - posRadB;
//			Real3 dist3 = Distance(posRadA, posRadB);
			Real3 dist3 = Modify_Local_PosB(posRadB, posRadA);
			Real d = length(dist3);
			if (d > RESOLUTION_LENGTH_MULT * paramsD.HSML)
				continue;

			Real4 rhoPresMuB = FETCH(sortedRhoPreMu, j);
//			// old version. When rigid-rigid contact used to be handled within fluid
//			if ((fabs(rhoPresMuB.w - rhoPresMuA.w) < .1)
//					&& rhoPresMuA.w > -.1) {
//				continue;
//			}
			if (rhoPresMuA.w > -.1 && rhoPresMuB.w > -.1) { // no rigid-rigid force
				continue;
			}

			modifyPressure(rhoPresMuB, dist3Alpha);
			Real3 velMasB = FETCH(sortedVelMas, j);
			if (rhoPresMuB.w > -.1) {
				int bceIndexB = gridMarkerIndex[j] - (numObjectsD.numFluidMarkers);
				if (!(bceIndexB >= 0 && bceIndexB < numObjectsD.numBoundaryMarkers + numObjectsD.numRigid_SphMarkers)) {
					printf("Error! bceIndex out of bound, collideD !\n");
				}
				rhoPresMuB = rhoPreMu_ModifiedBCE[bceIndexB];
				velMasB = velMas_ModifiedBCE[bceIndexB];
			}
			Real multViscosit = 1;
			Real4 derivVelRhoAB = mR4(0.0f);
			derivVelRhoAB = DifVelocityRho_implicit(dist3, d, posRadA, posRadB, velMasA,
					velMasB, rhoPresMuA, rhoPresMuB,
					multViscosit);
			derivVelRho += derivVelRhoAB;
		}
	}

	// ff1
	//	if (rhoPresMuA.w > 0) printf("force value %f %f %f\n", 1e20*derivV.x, 1e20*derivV.y, 1e20*derivV.z);
	return derivVelRho;
}
//--------------------------------------------------------------------------------------------------------------------------------


/**
 * @brief calcHashD
 * @details
 * 		 1. Get particle index. Determine by the block and thread we are in.
 * 		 2. From x,y,z position determine which bin it is in.
 * 		 3. Calculate hash from bin index.
 * 		 4. Store hash and particle index associated with it.
 *
 * @param gridMarkerHash
 * @param gridMarkerIndex
 * @param posRad
 * @param numAllMarkers
 */
__global__ void calcHashD(uint* gridMarkerHash,   // output
		uint* gridMarkerIndex,  // output
		Real3* posRad,          // input: positions
		uint numAllMarkers, volatile bool *isErrorD) {

	/* Calculate the index of where the particle is stored in posRad. */
	uint index = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;
	if (index >= numAllMarkers)
		return;

	Real3 p = posRad[index];

	if (!(isfinite(p.x) && isfinite(p.y) && isfinite(p.z))) {
		printf("Error! particle position is NAN: thrown from SDKCollisionSystem.cu, calcHashD !\n");
		*isErrorD = true;
		return;
	}

	/* Check particle is inside the domain. */
	Real3 boxCorner = paramsD.worldOrigin;
	if (p.x < boxCorner.x || p.y < boxCorner.y || p.z < boxCorner.z) {
		printf("Out of Min Boundary, point %f %f %f, boundary min: %f %f %f. Thrown from SDKCollisionSystem.cu, calcHashD !\n",
				p.x, p.y, p.z,boxCorner.x, boxCorner.y, boxCorner.z);
		*isErrorD = true;
		return;
	}
	boxCorner = paramsD.worldOrigin + paramsD.boxDims;
	if (p.x > boxCorner.x || p.y > boxCorner.y || p.z > boxCorner.z) {
		printf(
				"Out of max Boundary, point %f %f %f, boundary max: %f %f %f. Thrown from SDKCollisionSystem.cu, calcHashD !\n",
				p.x, p.y, p.z, boxCorner.x, boxCorner.y, boxCorner.z);
		*isErrorD = true;
		return;
	}

	/* Get x,y,z bin index in grid */
	int3 gridPos = calcGridPos(p);
	/* Calculate a hash from the bin index */
	uint hash = calcGridHash(gridPos);

	/* Store grid hash */
	gridMarkerHash[index] = hash;
	/* Store particle index associated to the hash we stored in gridMarkerHash */
	gridMarkerIndex[index] = index;
}

/**
 * @brief reorderDataAndFindCellStartD
 * @details See SDKCollisionSystem.cuh for more info
 */
__global__ void reorderDataAndFindCellStartD(uint* cellStart, // output: cell start index
		uint* cellEnd,        // output: cell end index
		Real3* sortedPosRad,  // output: sorted positions
		Real3* sortedVelMas,  // output: sorted velocities
		Real4* sortedRhoPreMu, uint* gridMarkerHash, // input: sorted grid hashes
		uint* gridMarkerIndex,      // input: sorted particle indices
		uint* mapOriginalToSorted, // mapOriginalToSorted[originalIndex] = originalIndex
		Real3* oldPosRad,           // input: sorted position array
		Real3* oldVelMas,           // input: sorted velocity array
		Real4* oldRhoPreMu, uint numAllMarkers) {
	extern __shared__ uint sharedHash[];  // blockSize + 1 elements
	/* Get the particle index the current thread is supposed to be looking at. */
	uint index = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;
	uint hash;
	/* handle case when no. of particles not multiple of block size */
	if (index < numAllMarkers) {
		hash = gridMarkerHash[index];
		/* Load hash data into shared memory so that we can look at neighboring particle's hash
		 * value without loading two hash values per thread
		 */
		sharedHash[threadIdx.x + 1] = hash;

		if (index > 0 && threadIdx.x == 0) {
			/* first thread in block must load neighbor particle hash */
			sharedHash[0] = gridMarkerHash[index - 1];
		}
	}

	__syncthreads();

	if (index < numAllMarkers) {
		/* If this particle has a different cell index to the previous particle then it must be
		 * the first particle in the cell, so store the index of this particle in the cell. As it
		 * isn't the first particle, it must also be the cell end of the previous particle's cell
		 */
		if (index == 0 || hash != sharedHash[threadIdx.x]) {
			cellStart[hash] = index;
			if (index > 0)
				cellEnd[sharedHash[threadIdx.x]] = index;
		}

		if (index == numAllMarkers - 1) {
			cellEnd[hash] = index + 1;
		}

		/* Now use the sorted index to reorder the pos and vel data */
		uint originalIndex = gridMarkerIndex[index];  // map sorted to original
		mapOriginalToSorted[index] = index;	// will be sorted outside. Alternatively, you could have mapOriginalToSorted[originalIndex] = index; without need to sort. But that is not thread safe
		Real3 posRad = FETCH(oldPosRad, originalIndex); // macro does either global read or texture fetch
		Real3 velMas = FETCH(oldVelMas, originalIndex); // see particles_kernel.cuh
		Real4 rhoPreMu = FETCH(oldRhoPreMu, originalIndex);

		if (!(isfinite(posRad.x) && isfinite(posRad.y)
				&& isfinite(posRad.z))) {
			printf("Error! particle position is NAN: thrown from SDKCollisionSystem.cu, reorderDataAndFindCellStartD !\n");
		}
		if (!(isfinite(velMas.x) && isfinite(velMas.y)
				&& isfinite(velMas.z))) {
			printf("Error! particle velocity is NAN: thrown from SDKCollisionSystem.cu, reorderDataAndFindCellStartD !\n");
		}
		if (!(isfinite(rhoPreMu.x) && isfinite(rhoPreMu.y)
				&& isfinite(rhoPreMu.z) && isfinite(rhoPreMu.w))) {
			printf("Error! particle rhoPreMu is NAN: thrown from SDKCollisionSystem.cu, reorderDataAndFindCellStartD !\n");
		}
		sortedPosRad[index] = posRad;
		sortedVelMas[index] = velMas;
		sortedRhoPreMu[index] = rhoPreMu;
	}
}
//--------------------------------------------------------------------------------------------------------------------------------
__global__ void new_BCE_VelocityPressure(
		Real3* velMas_ModifiedBCE,    		// input: sorted velocities
		Real4* rhoPreMu_ModifiedBCE,  		// input: sorted velocities
		Real3* sortedPosRad,                // input: sorted positions
		Real3* sortedVelMas,                // input: sorted velocities
		Real4* sortedRhoPreMu,
		uint* cellStart,
		uint* cellEnd,
		uint* mapOriginalToSorted,
		Real3* bceAcc,
		int2 updatePortion,
		volatile bool *isErrorD) {
	uint bceIndex = blockIdx.x * blockDim.x + threadIdx.x;
	uint sphIndex = bceIndex + updatePortion.x; // updatePortion = [start, end] index of the update portion
	if (sphIndex >= updatePortion.y) {
		return;
	}
	uint idA = mapOriginalToSorted[sphIndex];
	Real4 rhoPreMuA = FETCH(sortedRhoPreMu, idA);
	Real3 posRadA = FETCH(sortedPosRad, idA);
	Real3 velMasA = FETCH(sortedVelMas, idA);
	int isAffectedV = 0;
	int isAffectedP = 0;


	Real3 sumVW = mR3(0);
	Real sumWAll = 0;
	Real3 sumRhoRW = mR3(0);
	Real sumPW = 0;
	Real sumWFluid = 0;

	// get address in grid
	int3 gridPos = calcGridPos(posRadA);

	/// if (gridPos.x == paramsD.gridSize.x-1) printf("****aha %d %d\n", gridPos.x, paramsD.gridSize.x);

	// examine neighbouring cells
	for (int z = -1; z <= 1; z++) {
		for (int y = -1; y <= 1; y++) {
			for (int x = -1; x <= 1; x++) {
				int3 neighbourPos = gridPos + mI3(x, y, z);
				BCE_modification_Share(sumVW, sumWAll, sumRhoRW, sumPW, sumWFluid, isAffectedV, isAffectedP,
						neighbourPos, posRadA, sortedPosRad,
						sortedVelMas, sortedRhoPreMu, cellStart, cellEnd);
			}
		}
	}

	if (isAffectedV) {
		Real3 modifiedBCE_v = 2 * velMasA - sumVW / sumWAll;
		velMas_ModifiedBCE[bceIndex] = modifiedBCE_v;
	}
	if (isAffectedP) {
		// pressure
		Real3 a3 = mR3(0);
		if (fabs(rhoPreMuA.w) > 0) {  // rigid BCE
			int rigidBceIndex = sphIndex - numObjectsD.startRigidMarkers;
			if (rigidBceIndex < 0 || rigidBceIndex >= numObjectsD.numRigid_SphMarkers) {
				printf("Error! marker index out of bound: thrown from SDKCollisionSystem.cu, new_BCE_VelocityPressure !\n");
				*isErrorD = true;
				return;
			}
			a3 = bceAcc[rigidBceIndex];
		}
		Real pressure = (sumPW + dot(paramsD.gravity - a3, sumRhoRW))
				/ sumWFluid;  //(in fact:  (paramsD.gravity -
		// aW), but aW for moving rigids
		// is hard to calc. Assume aW is
		// zero for now
		Real density = InvEos(pressure);
		rhoPreMu_ModifiedBCE[bceIndex] = mR4(density, pressure, rhoPreMuA.z,
				rhoPreMuA.w);
	}
}
//--------------------------------------------------------------------------------------------------------------------------------
__global__ void collideD_implicit(Real4* sortedDerivVelRho_fsi_D,  // output: new velocity
		Real3* sortedPosRad,  // input: sorted positions
		Real3* sortedVelMas,  // input: sorted velocities
		Real4* sortedRhoPreMu,
		Real3* velMas_ModifiedBCE, Real4* rhoPreMu_ModifiedBCE, uint* gridMarkerIndex,
		uint* cellStart, uint* cellEnd, uint numAllMarkers, volatile bool *isErrorD) {

	uint index = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;
	if (index >= numAllMarkers)
		return;

	// read particle data from sorted arrays
	Real3 posRadA = FETCH(sortedPosRad, index);
	Real3 velMasA = FETCH(sortedVelMas, index);
	Real4 rhoPreMuA = FETCH(sortedRhoPreMu, index);


	// *** comment these couple of lines since we don't want the force on the rigid (or boundary) be influenced by ADAMi
	// *** method since it would cause large forces. ADAMI method is used only to calculate forces on the fluid markers (A)
	// *** near the boundary or rigid (B).
//	if (rhoPreMuA.w > -.1) {
//		int bceIndex = gridMarkerIndex[index] - (numObjectsD.numFluidMarkers);
//		if (!(bceIndex >= 0 && bceIndex < numObjectsD.numBoundaryMarkers + numObjectsD.numRigid_SphMarkers)) {
//			printf("Error! bceIndex out of bound, collideD !\n");
//			*isErrorD = true;
//		}
//		rhoPreMuA = rhoPreMu_ModifiedBCE[bceIndex];
//		velMasA = velMas_ModifiedBCE[bceIndex];
//	}

//	uint originalIndex = gridMarkerIndex[index];
	Real4 derivVelRho = sortedDerivVelRho_fsi_D[index];

	// get address in grid
	int3 gridPos = calcGridPos(posRadA);

	// examine neighbouring cells
	for (int x = -1; x <= 1; x++) {
		for (int y = -1; y <= 1; y++) {
			for (int z = -1; z <= 1; z++) {
				derivVelRho += collideCell_implicit(gridPos + mI3(x, y, z), index,
						posRadA, velMasA, rhoPreMuA, sortedPosRad,
						sortedVelMas, sortedRhoPreMu,
						velMas_ModifiedBCE, rhoPreMu_ModifiedBCE, gridMarkerIndex,
						cellStart, cellEnd);
			}
		}
	}

	// write new velocity back to original unsorted location
	// *** let's tweak a little bit :)
	if (!(isfinite(derivVelRho.x) && isfinite(derivVelRho.y)
			&& isfinite(derivVelRho.z) )) {
		printf("Error! particle derivVel is NAN: thrown from SDKCollisionSystem.cu, collideD !\n");
		*isErrorD = true;
	}
	if (!(isfinite(derivVelRho.w))) {
		printf("Error! particle derivRho is NAN: thrown from SDKCollisionSystem.cu, collideD !\n");
		*isErrorD = true;
	}
	sortedDerivVelRho_fsi_D[index] = derivVelRho;
}

//%%%%%%%%%%%%%%%%%%%%%%%%
//--------------------------------------------------------------------------------------------------------------------------------
//--------------------------------------------------------------------------------------------------------------------------------
// updates the fluid particles' properties, i.e. velocity, density, pressure, position
__global__ void UpdateFluidD_implicit(Real3* posRadD, Real3* velMasD,
		Real4* rhoPresMuD, Real4* derivVelRhoD, int2 updatePortion, Real dT, volatile bool *isErrorD) {
	uint index = blockIdx.x * blockDim.x + threadIdx.x;
	index += updatePortion.x; // updatePortion = [start, end] index of the update portion
	if (index >= updatePortion.y) {
		return;
	}
	Real4 derivVelRho = derivVelRhoD[index];
	Real4 rhoPresMu = rhoPresMuD[index];

	if (rhoPresMu.w < 0) {
		//-------------
		// ** position
		//-------------

		Real3 velMas = velMasD[index];
		Real3 posRad = posRadD[index];
		Real3 updatedPositon = posRad + velMas * dT;
		if (!(isfinite(updatedPositon.x) && isfinite(updatedPositon.y) && isfinite(updatedPositon.z))) {
			printf("Error! particle position is NAN: thrown from SDKCollisionSystem.cu, UpdateFluidDKernel !\n");
			*isErrorD = true;
			return;
		}
		posRadD[index] = updatedPositon;  // posRadD updated

		//-------------
		// ** velocity
		//-------------

		Real3 updatedVelocity = velMas + mR3(derivVelRho) * dT;



		if (!(isfinite(updatedVelocity.x) && isfinite(updatedVelocity.y) && isfinite(updatedVelocity.z))) {
			if (paramsD.enableAggressiveTweak) {
				updatedVelocity = mR3(0);
			} else {
				printf("Error! particle updatedVelocity is NAN: thrown from SDKCollisionSystem.cu, UpdateFluidDKernel !\n");
				*isErrorD = true;
				return;
			}
		}
		// 2*** let's tweak a little bit :)
		if (length(updatedVelocity)
				> paramsD.tweakMultV * paramsD.HSML / paramsD.dT
				&& paramsD.enableTweak) {
			updatedVelocity *= (paramsD.tweakMultV * paramsD.HSML / paramsD.dT)
					/ length(updatedVelocity);
		}
		// 2*** end tweak

		velMasD[index] = updatedVelocity;

	}
	// 3*** let's tweak a little bit :)
	if (!(isfinite(derivVelRho.w))) {
		if (paramsD.enableAggressiveTweak) {
			derivVelRho.w = 0;
		} else {
			printf("Error! particle derivVelRho.w is NAN: thrown from SDKCollisionSystem.cu, UpdateFluidDKernel !\n");
			*isErrorD = true;
			return;
		}
	}
	if (fabs(derivVelRho.w) > paramsD.tweakMultRho * paramsD.rho0 / paramsD.dT
			&& paramsD.enableTweak) {
		derivVelRho.w *= (paramsD.tweakMultRho * paramsD.rho0 / paramsD.dT)
				/ fabs(derivVelRho.w);  // to take care of the sign as well
	}
	// 2*** end tweak
	Real rho2 = rhoPresMu.x + derivVelRho.w * dT; // rho update. (i.e. rhoPresMu.x), still not wriiten to global matrix
	rhoPresMu.y = Eos(rho2, rhoPresMu.w);
	rhoPresMu.x = rho2;
	if (!(isfinite(rhoPresMu.x) && isfinite(rhoPresMu.y) && isfinite(rhoPresMu.z) && isfinite(rhoPresMu.w))) {
		printf("Error! particle rho pressure is NAN: thrown from SDKCollisionSystem.cu, UpdateFluidDKernel !\n");
		*isErrorD = true;
		return;
	}
	rhoPresMuD[index] = rhoPresMu;  // rhoPresMuD updated
}

/**
 * @brief Copies the sortedVelXSPH to velXSPH according to indexing
 * @details [long description]
 *
 * @param vel_XSPH_D
 * @param vel_XSPH_Sorted_D Pointer to new sorted vel_XSPH vector
 * @param m_dGridMarkerIndex List of indeces used to sort vel_XSPH_D
 */

__global__ void CopySorted_vXSPH_dVdRho_to_original_kernel(Real3* vel_XSPH_D,
		Real4* derivVelRhoD,
		Real3* vel_XSPH_Sorted_D, Real4* sortedDerivVelRho_fsi_D,
		uint* mapOriginalToSorted) {
	uint index = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;
	if (index >= numObjectsD.numAllMarkers)
		return;
	vel_XSPH_D[index] = vel_XSPH_Sorted_D[mapOriginalToSorted[index]];
	derivVelRhoD[index] = sortedDerivVelRho_fsi_D[mapOriginalToSorted[index]];
}

//--------------------------------------------------------------------------------------------------------------------------------
// updates the fluid particles' properties, i.e. velocity, density, pressure, position
__global__ void UpdateKernelBoundary(
		Real4* rhoPresMuD, Real4* derivVelRhoD, int2 updatePortion, Real dT) {
	uint index = blockIdx.x * blockDim.x + threadIdx.x;
	index += updatePortion.x; // updatePortion = [start, end] index of the update portion
	if (index >= updatePortion.y) {
		return;
	}

	Real4 derivVelRho = derivVelRhoD[index];
	Real4 rhoPresMu = rhoPresMuD[index];
	Real rho2 = rhoPresMu.x + derivVelRho.w * dT; // rho update. (i.e. rhoPresMu.x), still not wriiten to global matrix
	rhoPresMu.y = Eos(rho2, rhoPresMu.w);
	rhoPresMu.x = rho2;
	if (!(isfinite(rhoPresMu.x) && isfinite(rhoPresMu.y) && isfinite(rhoPresMu.z) && isfinite(rhoPresMu.w))) {
							printf("Error! particle rp is NAN: thrown from SDKCollisionSystem.cu, UpdateKernelBoundary !\n");
				}
	rhoPresMuD[index] = rhoPresMu;  // rhoPresMuD updated
}

//--------------------------------------------------------------------------------------------------------------------------------
// applies periodic BC along x
__global__ void ApplyPeriodicBoundaryXKernel(Real3* posRadD,
		Real4* rhoPresMuD) {
	uint index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= numObjectsD.numAllMarkers) {
		return;
	}
	Real4 rhoPresMu = rhoPresMuD[index];
	if (fabs(rhoPresMu.w) < .1) {
		return;
	}  // no need to do anything if it is a boundary particle
	Real3 posRad = posRadD[index];
	if (posRad.x > paramsD.cMax.x) {
		posRad.x -= (paramsD.cMax.x - paramsD.cMin.x);
		posRadD[index] = posRad;
		if (rhoPresMu.w < -.1) {
			rhoPresMu.y = rhoPresMu.y + paramsD.deltaPress.x;
			rhoPresMuD[index] = rhoPresMu;
		}
		return;
	}
	if (posRad.x < paramsD.cMin.x) {
		posRad.x += (paramsD.cMax.x - paramsD.cMin.x);
		posRadD[index] = posRad;
		if (rhoPresMu.w < -.1) {
			rhoPresMu.y = rhoPresMu.y - paramsD.deltaPress.x;
			rhoPresMuD[index] = rhoPresMu;
		}
		return;
	}
}
//--------------------------------------------------------------------------------------------------------------------------------
// applies periodic BC along y
__global__ void ApplyPeriodicBoundaryYKernel(Real3* posRadD,
		Real4* rhoPresMuD) {
	uint index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= numObjectsD.numAllMarkers) {
		return;
	}
	Real4 rhoPresMu = rhoPresMuD[index];
	if (fabs(rhoPresMu.w) < .1) {
		return;
	}  // no need to do anything if it is a boundary particle
	Real3 posRad = posRadD[index];
	if (posRad.y > paramsD.cMax.y) {
		posRad.y -= (paramsD.cMax.y - paramsD.cMin.y);
		posRadD[index] = posRad;
		if (rhoPresMu.w < -.1) {
			rhoPresMu.y = rhoPresMu.y + paramsD.deltaPress.y;
			rhoPresMuD[index] = rhoPresMu;
		}
		return;
	}
	if (posRad.y < paramsD.cMin.y) {
		posRad.y += (paramsD.cMax.y - paramsD.cMin.y);
		posRadD[index] = posRad;
		if (rhoPresMu.w < -.1) {
			rhoPresMu.y = rhoPresMu.y - paramsD.deltaPress.y;
			rhoPresMuD[index] = rhoPresMu;
		}
		return;
	}
}
//--------------------------------------------------------------------------------------------------------------------------------
// applies periodic BC along z
__global__ void ApplyPeriodicBoundaryZKernel(Real3* posRadD,
		Real4* rhoPresMuD) {
	uint index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index >= numObjectsD.numAllMarkers) {
		return;
	}
	Real4 rhoPresMu = rhoPresMuD[index];
	if (fabs(rhoPresMu.w) < .1) {
		return;
	}  // no need to do anything if it is a boundary particle
	Real3 posRad = posRadD[index];
	if (posRad.z > paramsD.cMax.z) {
		posRad.z -= (paramsD.cMax.z - paramsD.cMin.z);
		posRadD[index] = posRad;
		if (rhoPresMu.w < -.1) {
			rhoPresMu.y = rhoPresMu.y + paramsD.deltaPress.z;
			rhoPresMuD[index] = rhoPresMu;
		}
		return;
	}
	if (posRad.z < paramsD.cMin.z) {
		posRad.z += (paramsD.cMax.z - paramsD.cMin.z);
		posRadD[index] = posRad;
		if (rhoPresMu.w < -.1) {
			rhoPresMu.y = rhoPresMu.y - paramsD.deltaPress.z;
			rhoPresMuD[index] = rhoPresMu;
		}
		return;
	}
}

//%%%%%%%%%%%%%%%%%%%%%%%%
//--------------------------------------------------------------------------------------------------------------------------------
void allocateArray(void** devPtr, size_t size) {
	hipMalloc(devPtr, size);
}
//--------------------------------------------------------------------------------------------------------------------------------
void freeArray(void* devPtr) {
	hipFree(devPtr);
}

/**
 * @brief iDivUp
 * @details Round a / b to nearest higher integer value
 *
 * @param a numerator
 * @param b denominator
 *
 * @return ceil(a/b)
 */
uint iDivUp(uint a, uint b) {
	return (a % b != 0) ? (a / b + 1) : (a / b);
}

/**
 * @brief computeGridSize
 * @details Compute grid and thread block size for a given number of elements
 *
 * @param n Total number of elements. Each elements needs a thread to be computed
 * @param blockSize Number of threads per block.
 * @param numBlocks output
 * @param numThreads Output: number of threads per block
 */
void computeGridSize(uint n, uint blockSize, uint& numBlocks,
		uint& numThreads) {
	uint n2 = (n == 0) ? 1 : n;
	numThreads = min(blockSize, n2);
	numBlocks = iDivUp(n2, numThreads);
}

/**
 * @brief [brief description]
 * @details [long description]
 *
 * @param hostParams [description]
 * @param numObjects [description]
 */
void setParameters(SimParams* hostParams, NumberOfObjects* numObjects) {
	// copy parameters to constant memory
	hipMemcpyToSymbolAsync(HIP_SYMBOL(paramsD), hostParams, sizeof(SimParams));
	hipMemcpyToSymbolAsync(HIP_SYMBOL(numObjectsD), numObjects, sizeof(NumberOfObjects));
}

/**
 * @brief Wrapper function for calcHashD
 * @details See SDKCollisionSystem.cuh for more info
 */
void calcHash(thrust::device_vector<uint>& gridMarkerHash,
		thrust::device_vector<uint>& gridMarkerIndex,
		thrust::device_vector<Real3>& posRad,
		int numAllMarkers) {


	bool *isErrorH, *isErrorD;
	isErrorH = (bool *)malloc(sizeof(bool));
	hipMalloc((void**) &isErrorD, sizeof(bool));
	*isErrorH = false;
	hipMemcpy(isErrorD, isErrorH, sizeof(bool), hipMemcpyHostToDevice);
	//------------------------------------------------------------------------
	/* Is there a need to optimize the number of threads used at once? */
	uint numThreads, numBlocks;
	computeGridSize(numAllMarkers, 256, numBlocks, numThreads);
	/* Execute Kernel */
	calcHashD<<<numBlocks, numThreads>>>(U1CAST(gridMarkerHash),
			U1CAST(gridMarkerIndex), mR3CAST(posRad),
			numAllMarkers, isErrorD);

	/* Check for errors in kernel execution */
	hipDeviceSynchronize();
	cudaCheckError();
	//------------------------------------------------------------------------
	hipMemcpy(isErrorH, isErrorD, sizeof(bool), hipMemcpyDeviceToHost);
	if (*isErrorH == true) {
		throw std::runtime_error ("Error! program crashed in  calcHashD!\n");
	}
	hipFree(isErrorD);
	free(isErrorH);
}

/**
 * @brief Wrapper function for reorderDataAndFindCellStartD
 * @details
 * 		See SDKCollisionSystem.cuh for brief.
 */
void reorderDataAndFindCellStart(thrust::device_vector<uint>& cellStart,
		thrust::device_vector<uint>& cellEnd,
		thrust::device_vector<Real3>& sortedPosRad,
		thrust::device_vector<Real3>& sortedVelMas,
		thrust::device_vector<Real4>& sortedRhoPreMu,

		thrust::device_vector<uint>& gridMarkerHash,
		thrust::device_vector<uint>& gridMarkerIndex,

		thrust::device_vector<uint>& mapOriginalToSorted,

		thrust::device_vector<Real3>& oldPosRad,
		thrust::device_vector<Real3>& oldVelMas,
		thrust::device_vector<Real4>& oldRhoPreMu, uint numAllMarkers,
		uint numCells) {
	uint numThreads, numBlocks;
	computeGridSize(numAllMarkers, 256, numBlocks, numThreads); //?$ 256 is blockSize

	/* Set all cells to empty */
//	hipMemset(U1CAST(cellStart), 0xffffffff, numCells * sizeof(uint));
	thrust::fill(cellStart.begin(), cellStart.end(), 0);
	thrust::fill(cellEnd.begin(), cellEnd.end(), 0);

	//#if USE_TEX
	//#if 0
	//    cutilSafeCall(hipBindTexture(0, oldPosTex, oldPosRad, numAllMarkers*sizeof(Real4)));
	//    cutilSafeCall(hipBindTexture(0, oldVelTex, oldVelMas, numAllMarkers*sizeof(Real4)));
	//#endif

	uint smemSize = sizeof(uint) * (numThreads + 1);
	reorderDataAndFindCellStartD<<<numBlocks, numThreads, smemSize>>>(
			U1CAST(cellStart), U1CAST(cellEnd), mR3CAST(sortedPosRad),
			mR3CAST(sortedVelMas), mR4CAST(sortedRhoPreMu),
			U1CAST(gridMarkerHash), U1CAST(gridMarkerIndex),
			U1CAST(mapOriginalToSorted), mR3CAST(oldPosRad), mR3CAST(oldVelMas),
			mR4CAST(oldRhoPreMu), numAllMarkers);
	hipDeviceSynchronize();
	cudaCheckError()
	;

	// unroll sorted index to have the location of original particles in the sorted arrays
	thrust::device_vector<uint> dummyIndex = gridMarkerIndex;
	thrust::sort_by_key(dummyIndex.begin(), dummyIndex.end(),
			mapOriginalToSorted.begin());
	dummyIndex.clear();
	//#if USE_TEX
	//#if 0
	//    cutilSafeCall(hipUnbindTexture(oldPosTex));
	//    cutilSafeCall(hipUnbindTexture(oldVelTex));
	//#endif
}

//--------------------------------------------------------------------------------------------------------------------------------
void RecalcSortedVelocityPressure_BCE(
		thrust::device_vector<Real3>& velMas_ModifiedBCE,
		thrust::device_vector<Real4>& rhoPreMu_ModifiedBCE,
		const thrust::device_vector<Real3>& sortedPosRad,
		const thrust::device_vector<Real3>& sortedVelMas,
		const thrust::device_vector<Real4>& sortedRhoPreMu,
		const thrust::device_vector<uint>& cellStart,
		const thrust::device_vector<uint>& cellEnd,
		const thrust::device_vector<uint>& mapOriginalToSorted,
		const thrust::device_vector<Real3>& bceAcc,
		int2 updatePortion) {

	bool *isErrorH, *isErrorD;
	isErrorH = (bool *)malloc(sizeof(bool));
	hipMalloc((void**) &isErrorD, sizeof(bool));
	*isErrorH = false;
	hipMemcpy(isErrorD, isErrorH, sizeof(bool), hipMemcpyHostToDevice);
	//------------------------------------------------------------------------

	// thread per particle
	uint numThreads, numBlocks;
	computeGridSize(updatePortion.y - updatePortion.x, 64, numBlocks, numThreads);

	new_BCE_VelocityPressure<<<numBlocks, numThreads>>>(
			mR3CAST(velMas_ModifiedBCE),
			mR4CAST(rhoPreMu_ModifiedBCE),  // input: sorted velocities
			mR3CAST(sortedPosRad), mR3CAST(sortedVelMas),
			mR4CAST(sortedRhoPreMu), U1CAST(cellStart), U1CAST(cellEnd), U1CAST(mapOriginalToSorted),
			mR3CAST(bceAcc),
			updatePortion,
			isErrorD);

	hipDeviceSynchronize();
	cudaCheckError()

	//------------------------------------------------------------------------
	hipMemcpy(isErrorH, isErrorD, sizeof(bool), hipMemcpyDeviceToHost);
	if (*isErrorH == true) {
		throw std::runtime_error ("Error! program crashed in  new_BCE_VelocityPressure!\n");
	}
	hipFree(isErrorD);
	free(isErrorH);
}

//--------------------------------------------------------------------------------------------------------------------------------

/**
 * @brief Wrapper function for collide
 * @details
 * 		See SDKCollisionSystem.cuh for informaton on collide
 */
void collide_implicit(thrust::device_vector<Real4>& sortedDerivVelRho_fsi_D,
		thrust::device_vector<Real3>& sortedPosRad,
		thrust::device_vector<Real3>& sortedVelMas,
		thrust::device_vector<Real4>& sortedRhoPreMu,
		thrust::device_vector<Real3>& velMas_ModifiedBCE,
		thrust::device_vector<Real4>& rhoPreMu_ModifiedBCE,

		thrust::device_vector<uint>& gridMarkerIndex,
		thrust::device_vector<uint>& cellStart,
		thrust::device_vector<uint>& cellEnd, uint numAllMarkers, uint numCells,
		Real dT) {

	bool *isErrorH, *isErrorD;
	isErrorH = (bool *)malloc(sizeof(bool));
	hipMalloc((void**) &isErrorD, sizeof(bool));
	*isErrorH = false;
	hipMemcpy(isErrorD, isErrorH, sizeof(bool), hipMemcpyHostToDevice);
	//------------------------------------------------------------------------
	// thread per particle
	uint numThreads, numBlocks;
	computeGridSize(numAllMarkers, 64, numBlocks, numThreads);

	// execute the kernel
	collideD_implicit<<<numBlocks, numThreads>>>(mR4CAST(sortedDerivVelRho_fsi_D),
			mR3CAST(sortedPosRad), mR3CAST(sortedVelMas),
			mR4CAST(sortedRhoPreMu),
			mR3CAST(velMas_ModifiedBCE), mR4CAST(rhoPreMu_ModifiedBCE), U1CAST(gridMarkerIndex),
			U1CAST(cellStart), U1CAST(cellEnd),
			numAllMarkers, isErrorD);

	hipDeviceSynchronize();
	cudaCheckError();
	//------------------------------------------------------------------------
	hipMemcpy(isErrorH, isErrorD, sizeof(bool), hipMemcpyDeviceToHost);
	if (*isErrorH == true) {
		throw std::runtime_error ("Error! program crashed in  collideD!\n");
	}
	hipFree(isErrorD);
	free(isErrorH);
}

//--------------------------------------------------------------------------------------------------------------------------------
// updates the fluid particles by calling UpdateFluidD
void UpdateFluid_implicit(thrust::device_vector<Real3>& posRadD,
		thrust::device_vector<Real3>& velMasD,
		thrust::device_vector<Real4>& rhoPresMuD,
		thrust::device_vector<Real4>& derivVelRhoD,
		const thrust::host_vector<int4>& referenceArray, Real dT) {

//	int4 referencePortion = referenceArray[0];
//	if (referencePortion.z != -1) {
//		printf("error in UpdateFluid, accessing non fluid\n");
//		return;
//	}
//	int2 updatePortion = mI2(referencePortion);
	int2 updatePortion = mI2(0, referenceArray[referenceArray.size() - 1].y);
	// int2 updatePortion = mI2(referenceArray[0].x, referenceArray[0].y);

	bool *isErrorH, *isErrorD;
	isErrorH = (bool *)malloc(sizeof(bool));
	hipMalloc((void**) &isErrorD, sizeof(bool));
	*isErrorH = false;
	hipMemcpy(isErrorD, isErrorH, sizeof(bool), hipMemcpyHostToDevice);
	//------------------------
	uint nBlock_UpdateFluid, nThreads;
	computeGridSize(updatePortion.y - updatePortion.x, 128, nBlock_UpdateFluid,
			nThreads);
	UpdateFluidD_implicit<<<nBlock_UpdateFluid, nThreads>>>(mR3CAST(posRadD),
			mR3CAST(velMasD), mR4CAST(rhoPresMuD),
			mR4CAST(derivVelRhoD), updatePortion, dT, isErrorD);
	hipDeviceSynchronize();
	cudaCheckError();
	//------------------------
	hipMemcpy(isErrorH, isErrorD, sizeof(bool), hipMemcpyDeviceToHost);
	if (*isErrorH == true) {
		throw std::runtime_error ("Error! program crashed in  UpdateFluidD!\n");
	}
	hipFree(isErrorD);
	free(isErrorH);
}

//--------------------------------------------------------------------------------------------------------------------------------
// use invasive to avoid one extra copy. However, keep in mind that sorted is changed.
void CopySortedToOriginal_Invasive_R3(thrust::device_vector<Real3>& original,
		thrust::device_vector<Real3>& sorted,
		const thrust::device_vector<uint>& gridMarkerIndex) {
	thrust::device_vector<uint> dummyMarkerIndex = gridMarkerIndex;
	thrust::sort_by_key(dummyMarkerIndex.begin(), dummyMarkerIndex.end(),
			sorted.begin());
	dummyMarkerIndex.clear();
	thrust::copy(sorted.begin(), sorted.end(), original.begin());
}
//--------------------------------------------------------------------------------------------------------------------------------
void CopySortedToOriginal_NonInvasive_R3(thrust::device_vector<Real3>& original,
		thrust::device_vector<Real3>& sorted,
		const thrust::device_vector<uint>& gridMarkerIndex) {
	thrust::device_vector<Real3> dummySorted = sorted;
	CopySortedToOriginal_Invasive_R3(original, dummySorted, gridMarkerIndex);
}
//--------------------------------------------------------------------------------------------------------------------------------
// use invasive to avoid one extra copy. However, keep in mind that sorted is changed.
void CopySortedToOriginal_Invasive_R4(thrust::device_vector<Real4>& original,
		thrust::device_vector<Real4>& sorted,
		const thrust::device_vector<uint>& gridMarkerIndex) {
	thrust::device_vector<uint> dummyMarkerIndex = gridMarkerIndex;
	thrust::sort_by_key(dummyMarkerIndex.begin(), dummyMarkerIndex.end(),
			sorted.begin());
	dummyMarkerIndex.clear();
	thrust::copy(sorted.begin(), sorted.end(), original.begin());
}
//--------------------------------------------------------------------------------------------------------------------------------
void CopySortedToOriginal_NonInvasive_R4(thrust::device_vector<Real4>& original,
		thrust::device_vector<Real4>& sorted,
		const thrust::device_vector<uint>& gridMarkerIndex) {
	thrust::device_vector<Real4> dummySorted = sorted;
	CopySortedToOriginal_Invasive_R4(original, dummySorted, gridMarkerIndex);
}

//--------------------------------------------------------------------------------------------------------------------------------
// updates the fluid particles by calling UpdateBoundary
void UpdateBoundary(
		thrust::device_vector<Real4>& rhoPresMuD,
		thrust::device_vector<Real4>& derivVelRhoD,
		const thrust::host_vector<int4>& referenceArray, Real dT) {
	int4 referencePortion = referenceArray[1];
	if (referencePortion.z != 0) {
		printf("error in UpdateBoundary, accessing non boundary\n");
		return;
	}
	int2 updatePortion = mI2(referencePortion);

	uint nBlock_UpdateFluid, nThreads;
	computeGridSize(updatePortion.y - updatePortion.x, 128, nBlock_UpdateFluid,
			nThreads);
	UpdateKernelBoundary<<<nBlock_UpdateFluid, nThreads>>>(mR4CAST(rhoPresMuD), mR4CAST(derivVelRhoD),
			updatePortion, dT);
	hipDeviceSynchronize();
	cudaCheckError()
	;
}

/**
 * @brief ApplyBoundarySPH_Markers
 * @details
 * 		See SDKCollisionSystem.cuh for more info
 */
void ApplyBoundarySPH_Markers(thrust::device_vector<Real3>& posRadD,
		thrust::device_vector<Real4>& rhoPresMuD, int numAllMarkers) {
	uint nBlock_NumSpheres, nThreads_SphMarkers;
	computeGridSize(numAllMarkers, 256, nBlock_NumSpheres, nThreads_SphMarkers);
	ApplyPeriodicBoundaryXKernel<<<nBlock_NumSpheres, nThreads_SphMarkers>>>(
			mR3CAST(posRadD), mR4CAST(rhoPresMuD));
	hipDeviceSynchronize();
	cudaCheckError()
	;
	// these are useful anyway for out of bound particles
	ApplyPeriodicBoundaryYKernel<<<nBlock_NumSpheres, nThreads_SphMarkers>>>(
			mR3CAST(posRadD), mR4CAST(rhoPresMuD));
	hipDeviceSynchronize();
	cudaCheckError()
	;
	ApplyPeriodicBoundaryZKernel<<<nBlock_NumSpheres, nThreads_SphMarkers>>>(
			mR3CAST(posRadD), mR4CAST(rhoPresMuD));
	hipDeviceSynchronize();
	cudaCheckError()
	;

	//	SetOutputPressureToZero_X<<<nBlock_NumSpheres, nThreads_SphMarkers>>>(mR3CAST(posRadD), mR4CAST(rhoPresMuD));
	//    hipDeviceSynchronize();
	//    cudaCheckError();
}
